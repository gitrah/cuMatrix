#include "hip/hip_runtime.h"
/*
 * CuMatrixUnaryOps.cu
 *
 *      Author: reid
 */
#include "CuMatrix.h"
#include "caps.h"
#include "Kernels.h"

template<typename T> CuMatrix<T> CuMatrix<T>::negate() const {
	return unaryOp(Functory<T,negateUnaryOp>::pinch());
}

#ifdef  CuMatrix_Enable_KTS
template<typename T> template<template <typename> class UnaryOp> __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::unaryOp(UnaryOp<T> op, hipStream_t stream ) const
#else
template<typename T> template<int StateDim> __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::unaryOp(UnaryOpF<T,StateDim> op, hipStream_t stream ) const
#endif
{
	CuMatrix<T> res(m, n, true, true);
	if(checkDebug(debugUnaryOp)) {
		prlocf("in unaryOp(UnaryOp,...)\n");
		printShortString("unary op, src");
		res.printShortString("unary op, targ");
	}
	unaryOp(res, op, stream);
	return res;
}
#ifdef  CuMatrix_Enable_KTS

template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<absUnaryOp>(absUnaryOp<float>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<int> CuMatrix<int>::unaryOp<absUnaryOp>(absUnaryOp<int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<uint> CuMatrix<uint>::unaryOp<absUnaryOp>(absUnaryOp<uint>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<ulong> CuMatrix<ulong>::unaryOp<absUnaryOp>(absUnaryOp<ulong>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<long> CuMatrix<long>::unaryOp<absUnaryOp>(absUnaryOp<long>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<absUnaryOp>(absUnaryOp<double>, ihipStream_t*) const;

template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<powUnaryOp>(powUnaryOp<float>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<int> CuMatrix<int>::unaryOp<powUnaryOp>(powUnaryOp<int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<uint> CuMatrix<uint>::unaryOp<powUnaryOp>(powUnaryOp<uint>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<ulong> CuMatrix<ulong>::unaryOp<powUnaryOp>(powUnaryOp<ulong>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<long> CuMatrix<long>::unaryOp<powUnaryOp>(powUnaryOp<long>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<powUnaryOp>(powUnaryOp<double>, ihipStream_t*) const;

template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<expUnaryOp>(expUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<expUnaryOp>(expUnaryOp<double>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<translationUnaryOp>(translationUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<translationUnaryOp>(translationUnaryOp<double>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<ulong> CuMatrix<ulong>::unaryOp<translationUnaryOp>(translationUnaryOp<ulong>, hipStream_t) const;

template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<scaleUnaryOp>(scaleUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<scaleUnaryOp>(scaleUnaryOp<double>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<ulong> CuMatrix<ulong>::unaryOp<scaleUnaryOp>(scaleUnaryOp<ulong>, hipStream_t) const;

template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<subFromUnaryOp>(subFromUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<subFromUnaryOp>(subFromUnaryOp<double>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<ulong> CuMatrix<ulong>::unaryOp<subFromUnaryOp>(subFromUnaryOp<ulong>, hipStream_t) const;

template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<negateUnaryOp>(negateUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<negateUnaryOp>(negateUnaryOp<double>, hipStream_t) const;

template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<sigmoidUnaryOp>(sigmoidUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<sigmoidUnaryOp>(sigmoidUnaryOp<double>, hipStream_t) const;

template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<sigmoidGradientUnaryOp>(sigmoidGradientUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<sigmoidGradientUnaryOp>(sigmoidGradientUnaryOp<double>, hipStream_t) const;

template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<logUnaryOp>(logUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<logUnaryOp>(logUnaryOp<double>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<oneOverUnaryOp>(oneOverUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<oneOverUnaryOp>(oneOverUnaryOp<double>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<sqrtUnaryOp>(sqrtUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<sqrtUnaryOp>(sqrtUnaryOp<double>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<sqrUnaryOp>(sqrUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<sqrUnaryOp>(sqrUnaryOp<double>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<divSqrtUnaryOp>(divSqrtUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<divSqrtUnaryOp>(divSqrtUnaryOp<double>, hipStream_t) const;

template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<ltUnaryOp>(ltUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<ltUnaryOp>(ltUnaryOp<double>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<ulong> CuMatrix<ulong>::unaryOp<ltUnaryOp>(ltUnaryOp<ulong>, hipStream_t) const;

template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<lteUnaryOp>(lteUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<lteUnaryOp>(lteUnaryOp<double>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<ulong> CuMatrix<ulong>::unaryOp<lteUnaryOp>(lteUnaryOp<ulong>, hipStream_t) const;

template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<gtUnaryOp>(gtUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<gtUnaryOp>(gtUnaryOp<double>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<ulong> CuMatrix<ulong>::unaryOp<gtUnaryOp>(gtUnaryOp<ulong>, hipStream_t) const;

template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<gteUnaryOp>(gteUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<gteUnaryOp>(gteUnaryOp<double>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<ulong> CuMatrix<ulong>::unaryOp<gteUnaryOp>(gteUnaryOp<ulong>, hipStream_t) const;

template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp<eqUnaryOp>(eqUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp<eqUnaryOp>(eqUnaryOp<double>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<ulong> CuMatrix<ulong>::unaryOp<eqUnaryOp>(eqUnaryOp<ulong>, hipStream_t) const;

template __host__ CUDART_DEVICE CuMatrix<int> CuMatrix<int>::unaryOp<subFromUnaryOp>(subFromUnaryOp<int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<unsigned int> CuMatrix<unsigned int>::unaryOp<subFromUnaryOp>(subFromUnaryOp<unsigned int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<int> CuMatrix<int>::unaryOp<ltUnaryOp>(ltUnaryOp<int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<int> CuMatrix<int>::unaryOp<lteUnaryOp>(lteUnaryOp<int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<int> CuMatrix<int>::unaryOp<gtUnaryOp>(gtUnaryOp<int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<int> CuMatrix<int>::unaryOp<gteUnaryOp>(gteUnaryOp<int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<int> CuMatrix<int>::unaryOp<eqUnaryOp>(eqUnaryOp<int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<int> CuMatrix<int>::unaryOp<translationUnaryOp>(translationUnaryOp<int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<int> CuMatrix<int>::unaryOp<scaleUnaryOp>(scaleUnaryOp<int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<unsigned int> CuMatrix<unsigned int>::unaryOp<ltUnaryOp>(ltUnaryOp<unsigned int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<unsigned int> CuMatrix<unsigned int>::unaryOp<lteUnaryOp>(lteUnaryOp<unsigned int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<unsigned int> CuMatrix<unsigned int>::unaryOp<gtUnaryOp>(gtUnaryOp<unsigned int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<unsigned int> CuMatrix<unsigned int>::unaryOp<gteUnaryOp>(gteUnaryOp<unsigned int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<unsigned int> CuMatrix<unsigned int>::unaryOp<eqUnaryOp>(eqUnaryOp<unsigned int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<unsigned int> CuMatrix<unsigned int>::unaryOp<translationUnaryOp>(translationUnaryOp<unsigned int>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<unsigned int> CuMatrix<unsigned int>::unaryOp<scaleUnaryOp>(scaleUnaryOp<unsigned int>, ihipStream_t*) const;

template __host__ CUDART_DEVICE CuMatrix<long> CuMatrix<long>::unaryOp<scaleUnaryOp>(scaleUnaryOp<long>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<long> CuMatrix<long>::unaryOp<gteUnaryOp>(gteUnaryOp<long>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<long> CuMatrix<long>::unaryOp<translationUnaryOp>(translationUnaryOp<long>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<long> CuMatrix<long>::unaryOp<lteUnaryOp>(lteUnaryOp<long>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<long> CuMatrix<long>::unaryOp<ltUnaryOp>(ltUnaryOp<long>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<long> CuMatrix<long>::unaryOp<eqUnaryOp>(eqUnaryOp<long>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<long> CuMatrix<long>::unaryOp<gtUnaryOp>(gtUnaryOp<long>, ihipStream_t*) const;
template __host__ CUDART_DEVICE CuMatrix<long> CuMatrix<long>::unaryOp<subFromUnaryOp>(subFromUnaryOp<long>, ihipStream_t*) const;

#else
template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp(UnaryOpF<float,0>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp(UnaryOpF<double,0>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<long> CuMatrix<long>::unaryOp(UnaryOpF<long,0>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<ulong> CuMatrix<ulong>::unaryOp(UnaryOpF<ulong,0>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<int> CuMatrix<int>::unaryOp(UnaryOpF<int,0>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<uint> CuMatrix<uint>::unaryOp(UnaryOpF<uint,0>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp(UnaryOpF<float,1>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp(UnaryOpF<double,1>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<long> CuMatrix<long>::unaryOp(UnaryOpF<long,1>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<ulong> CuMatrix<ulong>::unaryOp(UnaryOpF<ulong,1>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<int> CuMatrix<int>::unaryOp(UnaryOpF<int,1>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<uint> CuMatrix<uint>::unaryOp(UnaryOpF<uint,1>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<float> CuMatrix<float>::unaryOp(UnaryOpF<float,2>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<double> CuMatrix<double>::unaryOp(UnaryOpF<double,2>, hipStream_t) const;
template __host__ CUDART_DEVICE CuMatrix<ulong> CuMatrix<ulong>::unaryOp(UnaryOpF<ulong,2>, hipStream_t) const;
#endif

#ifdef  CuMatrix_Enable_KTS
template<typename T> template<template <typename> class UnaryOp> __host__ CUDART_DEVICE void CuMatrix<T>::unaryOp(CuMatrix<T>& res, UnaryOp<T> op, hipStream_t stream) const
#else
template<typename T> template<int StateDim> __host__ CUDART_DEVICE void CuMatrix<T>::unaryOp(CuMatrix<T>& res, UnaryOpF<T,StateDim> op, hipStream_t stream) const
#endif
{
/*
	if(checkDebug(debugUnaryOp)) {
		flprintf("unaryOp tileCount %d lastMod %s\n", tiler.getTileCount(), b_util::modStr(lastMod));
	}
*/

	uint tileM, tileN, roff, coff;
	tiler.tileDims(tileM, tileN, tdRows);
	int tileCount = DIV_UP(m,tileM);
	DMatrix<T> d_A, d_Res;
	int lastGpu = ExecCaps::currDev();
	for(int i = 0; i < tileCount; i++) {
		if(checkDebug(debugFill))flprintf("tileM %d tileN %d tile %d lastGpu %u\n", tileM, tileN, i, lastGpu);
		if(checkDebug(debugFill))flprintf("roff %u coff %u\n",roff, coff);
		tiler.tileLike(d_A, roff, coff, tileM, tileN, i, tdRows, lastMod == mod_host, lastGpu, stream);
		if(checkDebug(debugFill))flprintf("after tiler.tileLike for tile %d; roff %u coff %u\n", i, roff, coff);
		lastGpu = res.tiler.tileLike(d_Res, roff, coff, tileM, tileN, i, tdRows, false,lastGpu, stream);
		if(checkDebug(debugFill))flprintf("after res.tiler.tileLike for tile %d; roff %u coff %u lastGpu %d\n", i, roff, coff, lastGpu);
		if(p == n) {
			unaryOpL( d_Res, d_A, op,stream);
		} else {
			if(checkDebug(debugUnaryOp)) {
				printf("invoking DMatrix version of unaryOp\n");
			}
			unaryOpDmL(d_Res, d_A, op, DefaultWidth2Height , stream);
		}
		res.tiler.syncTile(d_Res, roff, coff, stream);
	}
	if(checkDebug(debugUnaryOp)) {
		printDevArray(d_Res.elements,"d_Res",-1, MIN(10, m*n));
		printColoArray(res.elements,MIN(10, m*n));
	}

	//res.invalidateHost();
	res.lastMod = (tileCount>1) ? mod_host : mod_synced;
}
#ifdef  CuMatrix_Enable_KTS
template __host__ CUDART_DEVICE void CuMatrix<float>::unaryOp<approxInvSqrtUnaryOp>(CuMatrix<float>&, approxInvSqrtUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE void CuMatrix<double>::unaryOp<approxInvSqrtUnaryOp>(CuMatrix<double>&, approxInvSqrtUnaryOp<double>, hipStream_t) const;
template __host__ CUDART_DEVICE void CuMatrix<float>::unaryOp<slowInvSqrtUnaryOp>(CuMatrix<float>&, slowInvSqrtUnaryOp<float>, hipStream_t) const;
template __host__ CUDART_DEVICE void CuMatrix<double>::unaryOp<slowInvSqrtUnaryOp>(CuMatrix<double>&, slowInvSqrtUnaryOp<double>, hipStream_t) const;
#else
template __host__ CUDART_DEVICE void CuMatrix<float>::unaryOp(CuMatrix<float>&, UnaryOpF<float,0>, hipStream_t) const;
template __host__ CUDART_DEVICE void CuMatrix<double>::unaryOp(CuMatrix<double>&, UnaryOpF<double,0>, hipStream_t) const;
#endif

template<typename T> __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::sigmoid() const {
	return unaryOp(Functory<T,sigmoidUnaryOp>::pinch());
}

template<typename T> __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::sigmoidGradient() const {
	return unaryOp(Functory<T,sigmoidGradientUnaryOp>::pinch());
}

template<typename T> __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::log() const {
	return unaryOp(Functory<T,logUnaryOp>::pinch());
}

template<typename T> __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::ceil() const {
	return unaryOp(Functory<T,ceilUnaryOp>::pinch());
}

template<typename T> __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::floor() const {
	return unaryOp(Functory<T,floorUnaryOp>::pinch());
}


template<typename T> __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::oneOver() const {
	return unaryOp(Functory<T,oneOverUnaryOp>::pinch());
}

template<typename T> __host__ CUDART_DEVICE void CuMatrix<T>::setAll(int val) {
	assert(tiler.tileSize == tiler.m_size);
#ifndef __CUDA_ARCH__
	checkCudaErrors(hipMemset( tiler.currBuffer(), val, size));
#else
	memset(tiler.currBuffer(), val, size);
#endif
	lastMod = mod_device;
}

template<typename T> __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::exp() const {
	return unaryOp(Functory<T,expUnaryOp>::pinch());
}

template<typename T> __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::sqrt() const {
	return unaryOp(Functory<T,sqrtUnaryOp>::pinch());
}

template<typename T> __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::sqr() const {
	return unaryOp(Functory<T,sqrUnaryOp>::pinch());
}

template<typename T> __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::pow(T o) const {
	powUnaryOp<T> pf = Functory<T,powUnaryOp>::pinch(o);
	return unaryOp(pf);
}

template<typename T> __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::divSqrt(T divisor) const {
	divSqrtUnaryOp<T> dsf = Functory<T,divSqrtUnaryOp>::pinch(divisor);
	return unaryOp(dsf);
}

#ifdef  CuMatrix_Enable_KTS
template<typename T> template<template <typename> class BoolUnaryOp> __host__ CUDART_DEVICE bool CuMatrix<T>::all(BoolUnaryOp<T> op) const
{
	return gloloReduce(op, andBinaryOp<T>(), true);
}
#else
template<typename T> template <int StateDim> __host__ CUDART_DEVICE bool CuMatrix<T>::all(UnaryOpF<T,StateDim> op) const
{
	return gloloReduce(op, Functory<T, andBinaryOp>::pinch(), true);
}
#endif
#ifdef  CuMatrix_Enable_KTS
template __host__ CUDART_DEVICE bool CuMatrix<float>::all<almostEqUnaryOp>(almostEqUnaryOp<float>) const;
template __host__ CUDART_DEVICE bool CuMatrix<double>::all<almostEqUnaryOp>(almostEqUnaryOp<double>) const;
template __host__ CUDART_DEVICE bool CuMatrix<ulong>::all<almostEqUnaryOp>(almostEqUnaryOp<ulong>) const;
template __host__ CUDART_DEVICE bool CuMatrix<float>::all<ltUnaryOp>(ltUnaryOp<float>) const;
template __host__ CUDART_DEVICE bool CuMatrix<double>::all<ltUnaryOp>(ltUnaryOp<double>) const;
template __host__ CUDART_DEVICE bool CuMatrix<ulong>::all<ltUnaryOp>(ltUnaryOp<ulong>) const;
template __host__ CUDART_DEVICE bool CuMatrix<int>::all<almostEqUnaryOp>(almostEqUnaryOp<int>) const;
template __host__ CUDART_DEVICE bool CuMatrix<unsigned int>::all<almostEqUnaryOp>(almostEqUnaryOp<unsigned int>) const;
#else
template __host__ CUDART_DEVICE bool CuMatrix<float>::all<1>(UnaryOpF<float,1>) const;
template __host__ CUDART_DEVICE bool CuMatrix<double>::all<1>(UnaryOpF<double,1>) const;
template __host__ CUDART_DEVICE bool CuMatrix<long>::all<1>(UnaryOpF<long,1>) const;
template __host__ CUDART_DEVICE bool CuMatrix<ulong>::all<1>(UnaryOpF<ulong,1>) const;
template __host__ CUDART_DEVICE bool CuMatrix<float>::all<2>(UnaryOpF<float,2>) const;
template __host__ CUDART_DEVICE bool CuMatrix<double>::all<2>(UnaryOpF<double,2>) const;
template __host__ CUDART_DEVICE bool CuMatrix<int>::all<2>(UnaryOpF<int,2>) const;
template __host__ CUDART_DEVICE bool CuMatrix<uint>::all<2>(UnaryOpF<uint,2>) const;
template __host__ CUDART_DEVICE bool CuMatrix<long>::all<2>(UnaryOpF<long,2>) const;
template __host__ CUDART_DEVICE bool CuMatrix<ulong>::all<2>(UnaryOpF<ulong,2>) const;
#endif

#ifdef  CuMatrix_Enable_KTS
template<typename T> template<template <typename> class BoolUnaryOp> __host__ CUDART_DEVICE bool CuMatrix<T>::any(BoolUnaryOp<T> op) const
{
	return gloloReduce(op, orBinaryOp<T>(), false);
}
#else
template<typename T> template <int StateDim> __host__ CUDART_DEVICE bool CuMatrix<T>::any(UnaryOpF<T,StateDim> op) const
{
	return gloloReduce(op, Functory<T,orBinaryOp>::pinch(), false);
}
#endif
#ifdef  CuMatrix_Enable_KTS
template __host__ CUDART_DEVICE bool CuMatrix<float>::any<almostEqUnaryOp>(almostEqUnaryOp<float>) const;
template __host__ CUDART_DEVICE bool CuMatrix<double>::any<almostEqUnaryOp>(almostEqUnaryOp<double>) const;
template __host__ CUDART_DEVICE bool CuMatrix<ulong>::any<almostEqUnaryOp>(almostEqUnaryOp<ulong>) const;
template __host__ CUDART_DEVICE bool CuMatrix<float>::any<ltUnaryOp>(ltUnaryOp<float>) const;
template __host__ CUDART_DEVICE bool CuMatrix<double>::any<ltUnaryOp>(ltUnaryOp<double>) const;
template __host__ CUDART_DEVICE bool CuMatrix<ulong>::any<ltUnaryOp>(ltUnaryOp<ulong>) const;
#else
template __host__ CUDART_DEVICE bool CuMatrix<float>::any<1>(UnaryOpF<float,1>) const;
template __host__ CUDART_DEVICE bool CuMatrix<double>::any<1>(UnaryOpF<double,1>) const;
template __host__ CUDART_DEVICE bool CuMatrix<ulong>::any<1>(UnaryOpF<ulong,1>) const;
template __host__ CUDART_DEVICE bool CuMatrix<float>::any<2>(UnaryOpF<float,2>) const;
template __host__ CUDART_DEVICE bool CuMatrix<double>::any<2>(UnaryOpF<double,2>) const;
#endif

#ifdef  CuMatrix_Enable_KTS
template<typename T> template<template <typename> class BoolUnaryOp> __host__ CUDART_DEVICE bool CuMatrix<T>::none(	BoolUnaryOp<T> fn) const
#else
template<typename T> template <int StateDim> __host__ CUDART_DEVICE bool CuMatrix<T>::none(	UnaryOpF<T,StateDim> fn) const
#endif
{
	return !any(fn);
}
#ifdef  CuMatrix_Enable_KTS
template __host__ CUDART_DEVICE bool CuMatrix<float>::none<almostEqUnaryOp>(almostEqUnaryOp<float>) const;
template __host__ CUDART_DEVICE bool CuMatrix<double>::none<almostEqUnaryOp>(almostEqUnaryOp<double>) const;
template __host__ CUDART_DEVICE bool CuMatrix<ulong>::none<almostEqUnaryOp>(almostEqUnaryOp<ulong>) const;
template __host__ CUDART_DEVICE bool CuMatrix<float>::none<ltUnaryOp>(ltUnaryOp<float>) const;
template __host__ CUDART_DEVICE bool CuMatrix<double>::none<ltUnaryOp>(ltUnaryOp<double>) const;
template __host__ CUDART_DEVICE bool CuMatrix<ulong>::none<ltUnaryOp>(ltUnaryOp<ulong>) const;
template __host__ CUDART_DEVICE bool CuMatrix<float>::none<gtUnaryOp>(gtUnaryOp<float>) const;
template __host__ CUDART_DEVICE bool CuMatrix<double>::none<gtUnaryOp>(gtUnaryOp<double>) const;
template __host__ CUDART_DEVICE bool CuMatrix<ulong>::none<gtUnaryOp>(gtUnaryOp<ulong>) const;
#else
template __host__ CUDART_DEVICE bool CuMatrix<float>::none<1>(UnaryOpF<float,1>) const;
template __host__ CUDART_DEVICE bool CuMatrix<double>::none<1>(UnaryOpF<double,1>) const;
template __host__ CUDART_DEVICE bool CuMatrix<ulong>::none<1>(UnaryOpF<ulong,1>) const;
template __host__ CUDART_DEVICE bool CuMatrix<float>::none<2>(UnaryOpF<float,2>) const;
template __host__ CUDART_DEVICE bool CuMatrix<double>::none<2>(UnaryOpF<double,2>) const;
template __host__ CUDART_DEVICE bool CuMatrix<ulong>::none<2>(UnaryOpF<ulong,2>) const;
#endif

#ifdef  CuMatrix_Enable_KTS
template<typename T> template<template <typename> class BoolUnaryOp> __host__ CUDART_DEVICE long CuMatrix<T>::count(BoolUnaryOp<T> fn) const
{
	return gloloReduce(fn, plusBinaryOp<T>(), 0);
}
#else
template<typename T> template <int StateDim> __host__ CUDART_DEVICE long CuMatrix<T>::count( UnaryOpF<T,StateDim> fn) const
{
	return gloloReduce(fn, Functory<T, plusBinaryOp>::pinch(), 0);
}
#endif
#ifdef  CuMatrix_Enable_KTS
template __host__ CUDART_DEVICE long CuMatrix<float>::count<almostEqUnaryOp>(almostEqUnaryOp<float>) const;
template __host__ CUDART_DEVICE long CuMatrix<double>::count<almostEqUnaryOp>(almostEqUnaryOp<double>) const;
template __host__ CUDART_DEVICE long CuMatrix<ulong>::count<almostEqUnaryOp>(almostEqUnaryOp<ulong>) const;
template __host__ CUDART_DEVICE long CuMatrix<float>::count<notAlmostEqUnaryOp>(notAlmostEqUnaryOp<float>) const;
template __host__ CUDART_DEVICE long CuMatrix<double>::count<notAlmostEqUnaryOp>(notAlmostEqUnaryOp<double>) const;
template __host__ CUDART_DEVICE long CuMatrix<ulong>::count<notAlmostEqUnaryOp>(notAlmostEqUnaryOp<ulong>) const;
template __host__ CUDART_DEVICE long CuMatrix<float>::count<neqUnaryOp>(neqUnaryOp<float>) const;
template __host__ CUDART_DEVICE long CuMatrix<double>::count<neqUnaryOp>(neqUnaryOp<double>) const;
template __host__ CUDART_DEVICE long CuMatrix<ulong>::count<neqUnaryOp>(neqUnaryOp<ulong>) const;

template __host__ CUDART_DEVICE long CuMatrix<float>::count<ltUnaryOp>(ltUnaryOp<float>) const;
template __host__ CUDART_DEVICE long CuMatrix<double>::count<ltUnaryOp>(ltUnaryOp<double>) const;
template __host__ CUDART_DEVICE long CuMatrix<ulong>::count<ltUnaryOp>(ltUnaryOp<ulong>) const;
#else
template __host__ CUDART_DEVICE long CuMatrix<float>::count<1>(UnaryOpF<float,1>) const;
template __host__ CUDART_DEVICE long CuMatrix<double>::count<1>(UnaryOpF<double,1>) const;
template __host__ CUDART_DEVICE long CuMatrix<ulong>::count<1>(UnaryOpF<ulong,1>) const;
template __host__ CUDART_DEVICE long CuMatrix<float>::count<2>(UnaryOpF<float,2>) const;
template __host__ CUDART_DEVICE long CuMatrix<double>::count<2>(UnaryOpF<double,2>) const;
template __host__ CUDART_DEVICE long CuMatrix<ulong>::count<2>(UnaryOpF<ulong,2>) const;
#endif

#ifdef  CuMatrix_Enable_KTS
template<typename T> template<template <typename> class BoolUnaryOp> __host__ CUDART_DEVICE
IndexArray CuMatrix<T>::find(BoolUnaryOp<T> fn) const
#else
template<typename T> template <int StateDim> __host__ CUDART_DEVICE IndexArray CuMatrix<T>::find( UnaryOpF<T,StateDim> fn) const
#endif
{
	CuMatrix<T> m = unaryOp(fn);

	uint len = m.size/sizeof(T);
	int arraySize = 10;
	if(checkDebug(debugUnaryOp)) prlocf("creating intial idx array\n");
	uint* arry, *temp;
#ifdef __CUDA_ARCH__
	cherr(hipMalloc(&arry, arraySize * sizeof(uint)));
#else
	checkCudaError(hipHostAlloc(&arry, arraySize * sizeof(uint),0));
#endif
	int currIdx = 0;
	for(int i =0; i < len; i++ ) {
		if(m.get(i)) {
			flprintf("adding idx %d\n", i);
			arry[currIdx++] = i;
			if(currIdx == arraySize) {
				arraySize *= 2;
#ifdef __CUDA_ARCH__
	cherr(hipMalloc(&temp, arraySize * sizeof(uint)));
	cherr(hipMemcpyAsync(temp, arry, (currIdx -1) * sizeof(uint), hipMemcpyDeviceToDevice));
	cherr(hipDeviceSynchronize());
	cherr(hipFree(arry));
	arry = temp;
#else
	checkCudaError(hipHostAlloc(&temp, arraySize * sizeof(uint),0));
	cherr(hipMemcpy(temp, arry, (currIdx -1) * sizeof(uint), hipMemcpyHostToHost));
	if(checkDebug(debugDestr))flprintf("freeing host arry %p\n", arry);
	cherr(hipHostFree(arry));
	arry = temp;
#endif
			}
		} else {
			if(checkDebug(debugUnaryOp)) flprintf("skipping idx %d\n", i);
		}
	}
	if(currIdx < arraySize) {
		if(checkDebug(debugUnaryOp)) flprintf("shrinking idx array from %d to %d\n", arraySize, currIdx);
	}
	//arry = (uint*) realloc(arry, arraySize);
#ifdef __CUDA_ARCH__
	cherr(hipMalloc(&temp, arraySize * sizeof(uint)));
	cherr(hipMemcpyAsync(temp, arry, (currIdx -1)* sizeof(uint), hipMemcpyDeviceToDevice));
	cherr(hipDeviceSynchronize());
	cherr(hipFree(arry));
	arry = temp;
#else
	checkCudaError(hipHostAlloc(&temp, arraySize * sizeof(uint),0));
	cherr(hipMemcpy(temp, arry, (currIdx -1) * sizeof(uint), hipMemcpyHostToHost));
	if(checkDebug(debugDestr))flprintf("freeing host arry %p\n", arry);
	cherr(hipHostFree(arry));
	arry = temp;
#endif
	return IndexArray(arry, currIdx);
}
#ifdef  CuMatrix_Enable_KTS
template __host__ CUDART_DEVICE IndexArray CuMatrix<float>::find<ltUnaryOp>(ltUnaryOp<float>) const;
template __host__ CUDART_DEVICE IndexArray CuMatrix<double>::find<ltUnaryOp>(ltUnaryOp<double>) const;
#else
template __host__ CUDART_DEVICE IndexArray CuMatrix<float>::find<1>(UnaryOpF<float,1>) const;
template __host__ CUDART_DEVICE IndexArray CuMatrix<double>::find<1>(UnaryOpF<double,1>) const;
#endif

#ifdef  CuMatrix_Enable_KTS
template<typename T> template<template <typename> class BoolUnaryOp> __host__ CUDART_DEVICE
void CuMatrix<T>::findFirstN(IndexArray arry, BoolUnaryOp<T> op) const
#else
template<typename T> template <int StateDim> __host__ CUDART_DEVICE void CuMatrix<T>::findFirstN( IndexArray arry, UnaryOpF<T,StateDim> op) const
#endif
{
	CuMatrix<T> m = unaryOp(op);
	m.syncBuffers();

	uint len = m.size/sizeof(T);
	int currIdx = 0;
	for(int i =0; i < len; i++ ) {
		if(i == len -1 ){
			if(checkDebug(debugUnaryOp)) flprintf("lastIdx %d (+tiler.currBuffer() = %p)\n", i, i + tiler.currBuffer());
		}
		if(m.elements[i]) {
			if(checkDebug(debugUnaryOp)) flprintf("adding idx %d\n", i);
			if(currIdx < arry.count) {
				arry.indices[currIdx++] = i;
			} else {
				if(checkDebug(debugUnaryOp)) prlocf("exceeded capacity of indexarry; stopping\n");
				return;
			}
		} else {
		//	if(checkDebug(debugUnaryOp)) flprintf("skipping idx %d\n", i);
		}
	}
}
#ifdef  CuMatrix_Enable_KTS
template __host__ CUDART_DEVICE void CuMatrix<float>::findFirstN<ltUnaryOp>(IndexArray , ltUnaryOp<float>) const;
template __host__ CUDART_DEVICE void CuMatrix<double>::findFirstN<ltUnaryOp>(IndexArray, ltUnaryOp<double>) const;
template __host__ CUDART_DEVICE void CuMatrix<float>::findFirstN<gtUnaryOp>(IndexArray , gtUnaryOp<float>) const;
template __host__ CUDART_DEVICE void CuMatrix<double>::findFirstN<gtUnaryOp>(IndexArray, gtUnaryOp<double>) const;
template __host__ CUDART_DEVICE void CuMatrix<float>::findFirstN<notAlmostEqUnaryOp>(IndexArray , notAlmostEqUnaryOp<float>) const;
template __host__ CUDART_DEVICE void CuMatrix<double>::findFirstN<notAlmostEqUnaryOp>(IndexArray, notAlmostEqUnaryOp<double>) const;
template __host__ CUDART_DEVICE void CuMatrix<float>::findFirstN<neqUnaryOp>(IndexArray , neqUnaryOp<float>) const;
template __host__ CUDART_DEVICE void CuMatrix<double>::findFirstN<neqUnaryOp>(IndexArray, neqUnaryOp<double>) const;
template __host__ CUDART_DEVICE void CuMatrix<unsigned long>::findFirstN<ltUnaryOp>(IndexArray, ltUnaryOp<unsigned long>) const;
#else
template __host__ CUDART_DEVICE void CuMatrix<float>::findFirstN<1>(IndexArray , UnaryOpF<float,1>) const;
template __host__ CUDART_DEVICE void CuMatrix<ulong>::findFirstN<1>(IndexArray , UnaryOpF<ulong,1>) const;
template __host__ CUDART_DEVICE void CuMatrix<double>::findFirstN<1>(IndexArray, UnaryOpF<double,1>) const;
template __host__ CUDART_DEVICE void CuMatrix<float>::findFirstN<2>(IndexArray , UnaryOpF<float,2>) const;
template __host__ CUDART_DEVICE void CuMatrix<double>::findFirstN<2>(IndexArray, UnaryOpF<double,2>) const;
template __host__ CUDART_DEVICE void CuMatrix<ulong>::findFirstN<2>(IndexArray, UnaryOpF<ulong,2>) const;
#endif

#include "CuMatrixInster.cu"
