/*
 * testrowredux.cu
 *
 *  Created on: May 26, 2014
 *      Author: reid
 */
#include "tests.h"

#include "../CuMatrix.h"
#include "../util.h"
#include "../MatrixExceptions.h"
#include "../Maths.h"
#include "testKernels.h"

int launchDevInclisiveSum(uint fin) {
	if(fin == 0 || fin == 1) {
		return fin;
	}
	uint res = 0;
	uint* d_res;
	cherr(hipMalloc(&d_res,sizeof(uint)));
	inclusiveSum<<<1,1>>>( d_res, fin);
	cherr(hipMemcpy(&res,d_res,sizeof(uint), hipMemcpyDeviceToHost));
	cherr(hipFree(d_res));
	return res;
}
template int testReduceRows<float>::operator()(int argc, char const ** args) const;
template int testReduceRows<double>::operator()(int argc, char const ** args) const;
template int testReduceRows<ulong>::operator()(int argc, char const ** args) const;
template <typename T> int testReduceRows<T>::operator()(int argc, const char** args) const {
	outln("testReduceRows start " );
	plusBinaryOp<T> plus = Functory<T,plusBinaryOp>::pinch();
	int start = b_util::getStart(argc,args,1);
	for(int i = start; i < 65; i++ ) {
		outln("i " << i << ", inclusum " << launchDevInclisiveSum(i-1));
		CuMatrix<T> m1 = CuMatrix<T>::increasingColumns(0,1024,i);
		outln("m1 " << m1.syncBuffers());

		CuMatrix<T> resVec = CuMatrix<T>::zeros(m1.m,1);

		DMatrix<T> d_res, d_m1;
		m1.asDmatrix(d_m1);
		resVec.asDmatrix(d_res);

		CuMatrix<T>::reduceRows(d_res,d_m1,plus);

		T rvSum = resVec.sum();
		uint inclusum = launchDevInclisiveSum(i-1);
		outln("resVec " << resVec.syncBuffers() << "\nresVec.sum() " << rvSum);
		outln("should equals rows X inclusive sum (" << m1.m << " X " << inclusum << ")");
		assert(resVec.sum() ==  inclusum * d_res.m);
	}

	ulong len = 2 * Mega;
	CuMatrix<T> bigm1 = CuMatrix<T>::increasingColumns(0,len,64);
	outln("bigm1 " << bigm1.syncBuffers());
	T bigm1sum = bigm1.sum();
	T check  = 2l * Mega * 2016l;
	outln("bigm1.sum " << bigm1sum << ", check " << check);
	assert(bigm1sum  == check);

	CuMatrix<T> bigResVec = CuMatrix<T>::zeros(bigm1.m,1);

	DMatrix<T> d_bigres, d_bigm1;
	bigm1.asDmatrix(d_bigm1);
	bigResVec.asDmatrix(d_bigres);

	setCurrGpuDebugFlags( debugRedux,true,false);
	CuMatrix<T>::reduceRows(d_bigres,d_bigm1,plus);
	setCurrGpuDebugFlags( ~debugRedux,false,true);

	outln("bigResVec " << bigResVec.syncBuffers());
	assert(bigResVec.sum() == 2016 * d_bigres.m);


	/*

	CuMatrix<T> tinyOnes = CuMatrix<T>::ones(50,1);
	CuMatrix<T> tiny = tinyOnes |= (2 * tinyOnes);
	outln("tiny " << tiny.syncBuffers());
	outln("tiny col 0 sum " << tiny.reduceColumn(plus,0,0));
	outln("tiny col 1 sum " << tiny.reduceColumn(plus,0,1));

	CuMatrix<T> ones = CuMatrix<T>::ones(len,1);
	checkCudaError(hipGetLastError());
	T colOneSum = ones.columnSum(0);
	checkCudaError(hipGetLastError());
	outln("ones.colSum(0) " << colOneSum);

	T onesum = ones.sum();
	assert(colOneSum == onesum);
	outln("passed assert(colOneSum == onesum)");
*/
	return 0;
}


