#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <sstream>
#include "../caps.h"
#include "../debug.h"

#define tst_flprintf( format, ...) printf ( "[d]%s:%d %s " format, __FILE__, __LINE__, __func__,  __VA_ARGS__)
#define tst_prlocf(exp) 	printf( "[d]" __FILE__ "(%d): " exp, __LINE__)
__constant__ uint tst_debugFlags;
uint h_debugFlags;

using std::string;
using std::stringstream;
using std::cout;

inline __host__ __device__ bool tst_checkDebug(uint flags) {
#ifndef __CUDA_ARCH__
	return h_debugFlags & flags;
#else
	//#ifdef CuMatrix_DebugBuild
		return tst_debugFlags & flags;
	//#else
	//	return false;
	//#endif
#endif
}

void _setCurrGpuDebugFlags(uint flags, bool orThem, bool andThem,  hipStream_t stream ) {

	uint curr = flags;
	if(orThem) {
		tst_prlocf("copying DebugFlag fr device for or'n...\n");
		checkCudaErrors(hipMemcpyFromSymbol(&curr, HIP_SYMBOL(tst_debugFlags),sizeof(uint)));
		curr |= flags;
	} else if(andThem) {
		tst_prlocf("copying DebugFlag fr device fur and'n...\n");
		checkCudaErrors(hipMemcpyFromSymbol(&curr, HIP_SYMBOL(tst_debugFlags),sizeof(uint)));
		curr &= flags;
	}
	tst_prlocf("copying DebugFlag to device...\n");
	checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(tst_debugFlags),&curr,sizeof(uint),0,  hipMemcpyHostToDevice, stream));
	tst_prlocf("copied to device\n");
	h_debugFlags = curr;
}


void _setAllGpuDebugFlags(uint flags, bool orThem, bool andThem ) {
	tst_prlocf("_setAllGpuDebugFlags entre...\n");
	int devCount, currDev;
	checkCudaErrors(hipGetDeviceCount(&devCount));
	checkCudaErrors(hipGetDevice(&currDev));
	tst_flprintf("device count %d\n",devCount);
	tst_flprintf("curr device %d\n",currDev);

	hipStream_t *streams = (hipStream_t *) malloc(
			devCount * sizeof(hipStream_t));

	for(int i = 0; i < devCount;i++) {

		if(strstr("gtx980m", "750 Ti")) {
			tst_prlocf("not skipping sluggish 750 ti\n");
			//continue;
		}
		tst_flprintf("setting DbugFlags for device %s %d\n","gtx980m",i);

		ExecCaps_visitDevice(i);
		tst_flprintf("set device %d\n",i);
		checkCudaErrors(hipStreamCreateWithFlags(&(streams[i]), hipStreamNonBlocking));
		tst_prlocf("create stream\n");
		_setCurrGpuDebugFlags(flags,orThem,andThem, streams[i]);
		tst_prlocf("set gpu dbg flags\n");
	}

	for(int i = 0; i < devCount; i++) {
		tst_flprintf("synching stream for dev %d\n",i);
		checkCudaErrors(hipStreamSynchronize(streams[i]));
		checkCudaErrors(hipStreamDestroy(streams[i]));
	}

	ExecCaps_setDevice(currDev);
}
__host__ __device__ void tst_expNotation(char* buff, long val) {
	double factor = 1.;
	if (val >= Giga) {
		factor = 1. / Giga;
#ifndef __CUDA_ARCH__
		sprintf(buff, "%2.3gGb", val * factor);
#endif
	} else if (val >= Mega) {
		factor = 1. / Mega;
#ifndef __CUDA_ARCH__
		sprintf(buff, "%2.3gMb", val * factor);
#endif
	} else if (val >= Kilo) {
		factor = 1. / Kilo;
#ifndef __CUDA_ARCH__
		sprintf(buff, "%2.3gKb", val * factor);
#endif
	} else {
#ifndef __CUDA_ARCH__
		sprintf(buff, "%2.3gb", val * factor);
#endif
	}
}

string tst_expNotation(long val) {
	char buff[256];
	tst_expNotation(buff, val);
	stringstream ss;
	ss << buff;
	return ss.str();
}

__host__ void _allGpuMem(size_t* free, size_t* total) {
    int orgDev;
    checkCudaErrors(hipGetDevice(&orgDev));
    *free=0;
    *total=0;
    size_t lFree = 0, lTotal = 0;
	int devCnt;
	char buff[20];
	checkCudaErrors(hipGetDeviceCount(&devCnt));
	cout << "  ";
	assert(false);
	for(int i = 0; i < devCnt;i++) {
		ExecCaps_visitDevice(i);
		checkCudaErrors(hipMemGetInfo(&lFree, &lTotal));
		*free += lFree;
		*total += lTotal;
		sprintf(buff, " (%.2f%% used)", 100 * (1 - lFree * 1. / lTotal));
		if(tst_checkDebug(tst_debugFlags)) cout << "[" << i <<":  " <<  tst_expNotation(lFree) << " free /" << tst_expNotation(lTotal) << buff<< "] ";
	}
	cout << endl;
	ExecCaps_setDevice(orgDev);
}

double _usedMemRatio(bool allDevices) {
	size_t freeMemory, totalMemory;
	if(allDevices)
		_allGpuMem(&freeMemory, &totalMemory);
	else {
		assert(false);
		cout << "calling hipMemGetInfo\n";
		hipMemGetInfo(&freeMemory, &totalMemory);
		cout << "callied hipMemGetInfo\n";
	}
	int currDev;
	checkCudaErrors(hipGetDevice(&currDev));
	if (tst_debugFlags) {
		if(allDevices )
			cout << "\tallDev freeMemory " << freeMemory << ", total " << totalMemory << "\n";
		else
			cout << "\tdev " << currDev<< " freeMemory " << freeMemory << ", total " << totalMemory << "\n";
	}
	return 100 * (1 - freeMemory * 1. / totalMemory);
}

void _usedDmem(bool allDevices) {
	cout << "Memory " << _usedMemRatio(allDevices) << "% used\n";
}


int dmain(int argc, const char **argv) {
	uint localDbgFlags = 1 << 2 | 1 << 5;
	tst_flprintf("localDbgFlags %d\n",localDbgFlags);
	_usedDmem(true);
	_setAllGpuDebugFlags(localDbgFlags,false,false);
	tst_prlocf("set debug flags\n");
	_usedDmem(true);
}
