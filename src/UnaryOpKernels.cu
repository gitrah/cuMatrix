#include "hip/hip_runtime.h"
/*
 * UnaryOpKernels.cu
 *
 *  Created on: Oct 19, 2013
 *      Author: reid
 */



#include "util.h"
#include "caps.h"
#include "Kernels.h"
#include "CuFunctor.h"
#include "UnaryOpF_Gen.h"
#include <assert.h>

template<typename T, typename UnaryOp> __global__ void unaryOp1dKernel(
		T* trg, const T* src, UnaryOp op, ulong len) {
	ulong i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len) {
		trg[i] = op(src[i]);
	}
}

template<typename T, typename UnaryOp> __global__ void unaryOpDmKernel(
		DMatrix<T> trg, const DMatrix<T> src, UnaryOp op ) {
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.x + threadIdx.y;
	uint srcOff = y * src.p + x;
	uint trgOff = y * trg.p + x;
	for(int i = 0; i < blockDim.x; i+=blockDim.y) {
		if(x < src.n && y + i < src.m) {
			trg.elements[trgOff + i * trg.p] = op(src.elements[srcOff + i * src.p]);
		}
	}
}

#ifdef  CuMatrix_Enable_KTS
template<typename T, template <typename> class UnaryOp> __host__ CUDART_DEVICE void unaryOpL(DMatrix<T>& trg, const DMatrix<T>& src, UnaryOp<T> op, hipStream_t stream )
#else
template<typename T, int StateDim> __host__ CUDART_DEVICE void unaryOpL(DMatrix<T>& trg, const DMatrix<T>& src, UnaryOpF<T,StateDim> op, hipStream_t stream )
#endif
{
	int threads = 512;
	uint len = src.m * src.n;
	dim3 dBlocks, dThreads;
	b_util::vectorExecContext(threads, len, dBlocks, dThreads);
	unaryOp1dKernel<<<dBlocks,dThreads,0,stream>>>(trg.elements, src.elements, op, len);
}
#ifdef  CuMatrix_Enable_KTS
template __host__ CUDART_DEVICE void unaryOpL<float, approxInvSqrtUnaryOp>(DMatrix<float>&, DMatrix<float> const&, approxInvSqrtUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double, approxInvSqrtUnaryOp>(DMatrix<double>&, DMatrix<double> const&, approxInvSqrtUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<float, slowInvSqrtUnaryOp>(DMatrix<float>&, DMatrix<float> const&, slowInvSqrtUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double, slowInvSqrtUnaryOp>(DMatrix<double>&, DMatrix<double> const&, slowInvSqrtUnaryOp<double>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float, floorUnaryOp>(DMatrix<float>&, DMatrix<float> const&, floorUnaryOp<float>, ihipStream_t*);

template __host__ CUDART_DEVICE void unaryOpL<unsigned int, absUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, absUnaryOp<unsigned int>, ihipStream_t*);

template __host__ CUDART_DEVICE void unaryOpL<double, ceilUnaryOp>(DMatrix<double>&, DMatrix<double> const&, ceilUnaryOp<double>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, powUnaryOp>(DMatrix<long>&, DMatrix<long> const&, powUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, sigmoidGradientUnaryOp>(DMatrix<long>&, DMatrix<long> const&, sigmoidGradientUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, ceilUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, ceilUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, oneOverUnaryOp>(DMatrix<long>&, DMatrix<long> const&, oneOverUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, ltUnaryOp>(DMatrix<long>&, DMatrix<long> const&, ltUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, ceilUnaryOp>(DMatrix<long>&, DMatrix<long> const&, ceilUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, sqrtUnaryOp>(DMatrix<long>&, DMatrix<long> const&, sqrtUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned long, floorUnaryOp>(DMatrix<unsigned long>&, DMatrix<unsigned long> const&, floorUnaryOp<unsigned long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, gtUnaryOp>(DMatrix<long>&, DMatrix<long> const&, gtUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, ceilUnaryOp>(DMatrix<int>&, DMatrix<int> const&, ceilUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, negateUnaryOp>(DMatrix<long>&, DMatrix<long> const&, negateUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<double, floorUnaryOp>(DMatrix<double>&, DMatrix<double> const&, floorUnaryOp<double>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, divSqrtUnaryOp>(DMatrix<long>&, DMatrix<long> const&, divSqrtUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, logUnaryOp>(DMatrix<long>&, DMatrix<long> const&, logUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, gteUnaryOp>(DMatrix<long>&, DMatrix<long> const&, gteUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, floorUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, floorUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned long, ceilUnaryOp>(DMatrix<unsigned long>&, DMatrix<unsigned long> const&, ceilUnaryOp<unsigned long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, eqUnaryOp>(DMatrix<long>&, DMatrix<long> const&, eqUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, floorUnaryOp>(DMatrix<long>&, DMatrix<long> const&, floorUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, lteUnaryOp>(DMatrix<long>&, DMatrix<long> const&, lteUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, sqrUnaryOp>(DMatrix<long>&, DMatrix<long> const&, sqrUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<float, ceilUnaryOp>(DMatrix<float>&, DMatrix<float> const&, ceilUnaryOp<float>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, floorUnaryOp>(DMatrix<int>&, DMatrix<int> const&, floorUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, sigmoidUnaryOp>(DMatrix<long>&, DMatrix<long> const&, sigmoidUnaryOp<long>, ihipStream_t*);

template __host__ CUDART_DEVICE void unaryOpL<unsigned long, absUnaryOp>(DMatrix<unsigned long>&, DMatrix<unsigned long> const&, absUnaryOp<unsigned long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, absUnaryOp>(DMatrix<int>&, DMatrix<int> const&, absUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<double, absUnaryOp>(DMatrix<double>&, DMatrix<double> const&, absUnaryOp<double>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<float, absUnaryOp>(DMatrix<float>&, DMatrix<float> const&, absUnaryOp<float>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<long, absUnaryOp>(DMatrix<long>&, DMatrix<long> const&, absUnaryOp<long>, ihipStream_t*);

template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, absUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, absUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned long, absUnaryOp>(DMatrix<unsigned long>&, DMatrix<unsigned long> const&, absUnaryOp<unsigned long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, absUnaryOp>(DMatrix<int>&, DMatrix<int> const&, absUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<double, absUnaryOp>(DMatrix<double>&, DMatrix<double> const&, absUnaryOp<double>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<float, absUnaryOp>(DMatrix<float>&, DMatrix<float> const&, absUnaryOp<float>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, absUnaryOp>(DMatrix<long>&, DMatrix<long> const&, absUnaryOp<long>, int, ihipStream_t*);

template __host__ CUDART_DEVICE void unaryOpDmL<float, floorUnaryOp>(DMatrix<float>&, DMatrix<float> const&, floorUnaryOp<float>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<double, ceilUnaryOp>(DMatrix<double>&, DMatrix<double> const&, ceilUnaryOp<double>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, powUnaryOp>(DMatrix<long>&, DMatrix<long> const&, powUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, sigmoidGradientUnaryOp>(DMatrix<long>&, DMatrix<long> const&, sigmoidGradientUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, scaleUnaryOp>(DMatrix<long>&, DMatrix<long> const&, scaleUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, ceilUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, ceilUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, oneOverUnaryOp>(DMatrix<long>&, DMatrix<long> const&, oneOverUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, ltUnaryOp>(DMatrix<long>&, DMatrix<long> const&, ltUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, ceilUnaryOp>(DMatrix<long>&, DMatrix<long> const&, ceilUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, translationUnaryOp>(DMatrix<long>&, DMatrix<long> const&, translationUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, sqrtUnaryOp>(DMatrix<long>&, DMatrix<long> const&, sqrtUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned long, floorUnaryOp>(DMatrix<unsigned long>&, DMatrix<unsigned long> const&, floorUnaryOp<unsigned long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, gtUnaryOp>(DMatrix<long>&, DMatrix<long> const&, gtUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, ceilUnaryOp>(DMatrix<int>&, DMatrix<int> const&, ceilUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, negateUnaryOp>(DMatrix<long>&, DMatrix<long> const&, negateUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<double, floorUnaryOp>(DMatrix<double>&, DMatrix<double> const&, floorUnaryOp<double>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, divSqrtUnaryOp>(DMatrix<long>&, DMatrix<long> const&, divSqrtUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, logUnaryOp>(DMatrix<long>&, DMatrix<long> const&, logUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, gteUnaryOp>(DMatrix<long>&, DMatrix<long> const&, gteUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, floorUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, floorUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, expUnaryOp>(DMatrix<long>&, DMatrix<long> const&, expUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned long, ceilUnaryOp>(DMatrix<unsigned long>&, DMatrix<unsigned long> const&, ceilUnaryOp<unsigned long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, eqUnaryOp>(DMatrix<long>&, DMatrix<long> const&, eqUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, floorUnaryOp>(DMatrix<long>&, DMatrix<long> const&, floorUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, lteUnaryOp>(DMatrix<long>&, DMatrix<long> const&, lteUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, sqrUnaryOp>(DMatrix<long>&, DMatrix<long> const&, sqrUnaryOp<long>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<float, ceilUnaryOp>(DMatrix<float>&, DMatrix<float> const&, ceilUnaryOp<float>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, floorUnaryOp>(DMatrix<int>&, DMatrix<int> const&, floorUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, sigmoidUnaryOp>(DMatrix<long>&, DMatrix<long> const&, sigmoidUnaryOp<long>, int, ihipStream_t*);

template __host__ CUDART_DEVICE void unaryOpL<float,expUnaryOp>(DMatrix<float>&, const DMatrix<float>&, expUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,expUnaryOp>(DMatrix<double>&, const DMatrix<double>&, expUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<long,expUnaryOp>(DMatrix<long>&, const DMatrix<long>&, expUnaryOp<long>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,expUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, expUnaryOp<ulong>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,translationUnaryOp>(DMatrix<float>&, const DMatrix<float>&, translationUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,translationUnaryOp>(DMatrix<double>&, const DMatrix<double>&, translationUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<long,translationUnaryOp>(DMatrix<long>&, const DMatrix<long>&, translationUnaryOp<long>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,translationUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, translationUnaryOp<ulong>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,scaleUnaryOp>(DMatrix<float>&, const DMatrix<float>&, scaleUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,scaleUnaryOp>(DMatrix<double>&, const DMatrix<double>&, scaleUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<long,scaleUnaryOp>(DMatrix<long>&, const DMatrix<long>&, scaleUnaryOp<long>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,scaleUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, scaleUnaryOp<ulong>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,subFromUnaryOp>(DMatrix<float>&, const DMatrix<float>&, subFromUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,subFromUnaryOp>(DMatrix<double>&, const DMatrix<double>&, subFromUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,subFromUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, subFromUnaryOp<ulong>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,negateUnaryOp>(DMatrix<float>&, const DMatrix<float>&, negateUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,negateUnaryOp>(DMatrix<double>&, const DMatrix<double>&, negateUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,negateUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, negateUnaryOp<ulong>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,sigmoidUnaryOp>(DMatrix<float>&, const DMatrix<float>&, sigmoidUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,sigmoidUnaryOp>(DMatrix<double>&, const DMatrix<double>&, sigmoidUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,sigmoidUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, sigmoidUnaryOp<ulong>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,sigmoidGradientUnaryOp>(DMatrix<float>&, const DMatrix<float>&, sigmoidGradientUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,sigmoidGradientUnaryOp>(DMatrix<double>&, const DMatrix<double>&, sigmoidGradientUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,sigmoidGradientUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, sigmoidGradientUnaryOp<ulong>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,logUnaryOp>(DMatrix<float>&, const DMatrix<float>&, logUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,logUnaryOp>(DMatrix<double>&, const DMatrix<double>&, logUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,logUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, logUnaryOp<ulong>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<float,oneOverUnaryOp>(DMatrix<float>&, const DMatrix<float>&, oneOverUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,oneOverUnaryOp>(DMatrix<double>&, const DMatrix<double>&, oneOverUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,oneOverUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, oneOverUnaryOp<ulong>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<float,sqrtUnaryOp>(DMatrix<float>&, const DMatrix<float>&, sqrtUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,sqrtUnaryOp>(DMatrix<double>&, const DMatrix<double>&, sqrtUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,sqrtUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, sqrtUnaryOp<ulong>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<float,sqrUnaryOp>(DMatrix<float>&, const DMatrix<float>&, sqrUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,sqrUnaryOp>(DMatrix<double>&, const DMatrix<double>&, sqrUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,sqrUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, sqrUnaryOp<ulong>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<float,powUnaryOp>(DMatrix<float>&, const DMatrix<float>&, powUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,powUnaryOp>(DMatrix<double>&, const DMatrix<double>&, powUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,powUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, powUnaryOp<ulong>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<float,divSqrtUnaryOp>(DMatrix<float>&, const DMatrix<float>&, divSqrtUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,divSqrtUnaryOp>(DMatrix<double>&, const DMatrix<double>&, divSqrtUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,divSqrtUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, divSqrtUnaryOp<ulong>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,ltUnaryOp>(DMatrix<float>&, const DMatrix<float>&, ltUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,ltUnaryOp>(DMatrix<double>&, const DMatrix<double>&, ltUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,ltUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, ltUnaryOp<ulong>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,lteUnaryOp>(DMatrix<float>&, const DMatrix<float>&, lteUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,lteUnaryOp>(DMatrix<double>&, const DMatrix<double>&, lteUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,lteUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, lteUnaryOp<ulong>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,gtUnaryOp>(DMatrix<float>&, const DMatrix<float>&, gtUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,gtUnaryOp>(DMatrix<double>&, const DMatrix<double>&, gtUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,gtUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, gtUnaryOp<ulong>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,gteUnaryOp>(DMatrix<float>&, const DMatrix<float>&, gteUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,gteUnaryOp>(DMatrix<double>&, const DMatrix<double>&, gteUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,gteUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, gteUnaryOp<ulong>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,eqUnaryOp>(DMatrix<float>&, const DMatrix<float>&, eqUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,eqUnaryOp>(DMatrix<double>&, const DMatrix<double>&, eqUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,eqUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, eqUnaryOp<ulong>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,notAlmostEqUnaryOp>(DMatrix<float>&, const DMatrix<float>&, notAlmostEqUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,notAlmostEqUnaryOp>(DMatrix<double>&, const DMatrix<double>&, notAlmostEqUnaryOp<double>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<float,neqUnaryOp>(DMatrix<float>&, const DMatrix<float>&, neqUnaryOp<float>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,neqUnaryOp>(DMatrix<double>&, const DMatrix<double>&, neqUnaryOp<double>, ihipStream_t *);


template __host__ CUDART_DEVICE void unaryOpL<int, negateUnaryOp>(DMatrix<int>&, DMatrix<int> const&, negateUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, sigmoidUnaryOp>(DMatrix<int>&, DMatrix<int> const&, sigmoidUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, sigmoidGradientUnaryOp>(DMatrix<int>&, DMatrix<int> const&, sigmoidGradientUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, logUnaryOp>(DMatrix<int>&, DMatrix<int> const&, logUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, oneOverUnaryOp>(DMatrix<int>&, DMatrix<int> const&, oneOverUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, expUnaryOp>(DMatrix<int>&, DMatrix<int> const&, expUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, sqrtUnaryOp>(DMatrix<int>&, DMatrix<int> const&, sqrtUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, sqrUnaryOp>(DMatrix<int>&, DMatrix<int> const&, sqrUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, powUnaryOp>(DMatrix<int>&, DMatrix<int> const&, powUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, divSqrtUnaryOp>(DMatrix<int>&, DMatrix<int> const&, divSqrtUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, negateUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, negateUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, sigmoidUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, sigmoidUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, sigmoidGradientUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, sigmoidGradientUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, logUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, logUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, oneOverUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, oneOverUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, expUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, expUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, sqrtUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, sqrtUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, sqrUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, sqrUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, powUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, powUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, divSqrtUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, divSqrtUnaryOp<unsigned int>, ihipStream_t*);

template __host__ CUDART_DEVICE void unaryOpL<int, scaleUnaryOp>(DMatrix<int>&, DMatrix<int> const&, scaleUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, scaleUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, scaleUnaryOp<unsigned int>, ihipStream_t*);

template __host__ CUDART_DEVICE void unaryOpL<int, subFromUnaryOp>(DMatrix<int>&, DMatrix<int> const&, subFromUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, ltUnaryOp>(DMatrix<int>&, DMatrix<int> const&, ltUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, lteUnaryOp>(DMatrix<int>&, DMatrix<int> const&, lteUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, gtUnaryOp>(DMatrix<int>&, DMatrix<int> const&, gtUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, gteUnaryOp>(DMatrix<int>&, DMatrix<int> const&, gteUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, eqUnaryOp>(DMatrix<int>&, DMatrix<int> const&, eqUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<int, translationUnaryOp>(DMatrix<int>&, DMatrix<int> const&, translationUnaryOp<int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, subFromUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, subFromUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, ltUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, ltUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, lteUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, lteUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, gtUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, gtUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, gteUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, gteUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, eqUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, eqUnaryOp<unsigned int>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpL<unsigned int, translationUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, translationUnaryOp<unsigned int>, ihipStream_t*);

template __host__ CUDART_DEVICE void unaryOpL<long, subFromUnaryOp>(DMatrix<long>&, DMatrix<long> const&, subFromUnaryOp<long>, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<long, subFromUnaryOp>(DMatrix<long>&, DMatrix<long> const&, subFromUnaryOp<long>, int, ihipStream_t*);


#else
template __host__ CUDART_DEVICE void unaryOpL<float,0>(DMatrix<float>&, const DMatrix<float>&, UnaryOpF<float,0>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,0>(DMatrix<double>&, const DMatrix<double>&, UnaryOpF<double,0>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<long,0>(DMatrix<long>&, const DMatrix<long>&, UnaryOpF<long,0>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,0>(DMatrix<ulong>&, const DMatrix<ulong>&, UnaryOpF<ulong,0>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<int,0>(DMatrix<int>&, const DMatrix<int>&, UnaryOpF<int,0>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<uint,0>(DMatrix<uint>&, const DMatrix<uint>&, UnaryOpF<uint,0>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,1>(DMatrix<float>&, const DMatrix<float>&, UnaryOpF<float,1>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,1>(DMatrix<double>&, const DMatrix<double>&, UnaryOpF<double,1>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<long,1>(DMatrix<long>&, const DMatrix<long>&, UnaryOpF<long,1>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,1>(DMatrix<ulong>&, const DMatrix<ulong>&, UnaryOpF<ulong,1>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<int,1>(DMatrix<int>&, const DMatrix<int>&, UnaryOpF<int,1>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<uint,1>(DMatrix<uint>&, const DMatrix<uint>&, UnaryOpF<uint,1>, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpL<float,2>(DMatrix<float>&, const DMatrix<float>&, UnaryOpF<float,2>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<double,2>(DMatrix<double>&, const DMatrix<double>&, UnaryOpF<double,2>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<long,2>(DMatrix<long>&, const DMatrix<long>&, UnaryOpF<long,2>, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpL<ulong,2>(DMatrix<ulong>&, const DMatrix<ulong>&, UnaryOpF<ulong,2>, ihipStream_t *);
#endif


#ifdef  CuMatrix_Enable_KTS
template<typename T, template <typename> class UnaryOp> __host__ CUDART_DEVICE void unaryOpDmL(DMatrix<T>& trg, const DMatrix<T>& src, UnaryOp<T> op, int w2h, hipStream_t stream )
#else
template<typename T, int StateDim> __host__ CUDART_DEVICE void unaryOpDmL(DMatrix<T>& trg, const DMatrix<T>& src, UnaryOpF<T,StateDim> op, int w2h, hipStream_t stream )
#endif
{
	assert( trg.m >= src.m && trg.n >= src.n );
	int blockW = UNOP_BLOCK_SIZE;
	dim3 block(blockW,blockW/w2h);
    dim3 grid(DIV_UP(src.n,blockW), DIV_UP(src.m,blockW));
    if(checkDebug(debugExec)) { printf("unaryOpDmL grid "); b_util::prd3(grid); printf(" of block " );  b_util::prd3(block);}
    unaryOpDmKernel<<<grid,block,0,stream>>>(trg, src, op);
}
#ifdef  CuMatrix_Enable_KTS
template __host__ CUDART_DEVICE void unaryOpDmL<float, approxInvSqrtUnaryOp>(DMatrix<float>&, DMatrix<float> const&, approxInvSqrtUnaryOp<float>, int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double, approxInvSqrtUnaryOp>(DMatrix<double>&, DMatrix<double> const&, approxInvSqrtUnaryOp<double>, int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<float, slowInvSqrtUnaryOp>(DMatrix<float>&, DMatrix<float> const&, slowInvSqrtUnaryOp<float>, int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double, slowInvSqrtUnaryOp>(DMatrix<double>&, DMatrix<double> const&, slowInvSqrtUnaryOp<double>, int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,expUnaryOp>(DMatrix<float>&, const DMatrix<float>&, expUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,expUnaryOp>(DMatrix<double>&, const DMatrix<double>&, expUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,expUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, expUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,translationUnaryOp>(DMatrix<float>&, const DMatrix<float>&, translationUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,translationUnaryOp>(DMatrix<double>&, const DMatrix<double>&, translationUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,translationUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, translationUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,scaleUnaryOp>(DMatrix<float>&, const DMatrix<float>&, scaleUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,scaleUnaryOp>(DMatrix<double>&, const DMatrix<double>&, scaleUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,scaleUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, scaleUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,subFromUnaryOp>(DMatrix<float>&, const DMatrix<float>&, subFromUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,subFromUnaryOp>(DMatrix<double>&, const DMatrix<double>&, subFromUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,subFromUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, subFromUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,negateUnaryOp>(DMatrix<float>&, const DMatrix<float>&, negateUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,negateUnaryOp>(DMatrix<double>&, const DMatrix<double>&, negateUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,negateUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, negateUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,sigmoidUnaryOp>(DMatrix<float>&, const DMatrix<float>&, sigmoidUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,sigmoidUnaryOp>(DMatrix<double>&, const DMatrix<double>&, sigmoidUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,sigmoidUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, sigmoidUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,sigmoidGradientUnaryOp>(DMatrix<float>&, const DMatrix<float>&, sigmoidGradientUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,sigmoidGradientUnaryOp>(DMatrix<double>&, const DMatrix<double>&, sigmoidGradientUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,sigmoidGradientUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, sigmoidGradientUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,logUnaryOp>(DMatrix<float>&, const DMatrix<float>&, logUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,logUnaryOp>(DMatrix<double>&, const DMatrix<double>&, logUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,logUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, logUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,oneOverUnaryOp>(DMatrix<float>&, const DMatrix<float>&, oneOverUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,oneOverUnaryOp>(DMatrix<double>&, const DMatrix<double>&, oneOverUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,oneOverUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, oneOverUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,sqrtUnaryOp>(DMatrix<float>&, const DMatrix<float>&, sqrtUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,sqrtUnaryOp>(DMatrix<double>&, const DMatrix<double>&, sqrtUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,sqrtUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, sqrtUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,sqrUnaryOp>(DMatrix<float>&, const DMatrix<float>&, sqrUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,sqrUnaryOp>(DMatrix<double>&, const DMatrix<double>&, sqrUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,sqrUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, sqrUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,powUnaryOp>(DMatrix<float>&, const DMatrix<float>&, powUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,powUnaryOp>(DMatrix<double>&, const DMatrix<double>&, powUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,powUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, powUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,divSqrtUnaryOp>(DMatrix<float>&, const DMatrix<float>&, divSqrtUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,divSqrtUnaryOp>(DMatrix<double>&, const DMatrix<double>&, divSqrtUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,divSqrtUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, divSqrtUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,ltUnaryOp>(DMatrix<float>&, const DMatrix<float>&, ltUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,ltUnaryOp>(DMatrix<double>&, const DMatrix<double>&, ltUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,ltUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, ltUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,lteUnaryOp>(DMatrix<float>&, const DMatrix<float>&, lteUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,lteUnaryOp>(DMatrix<double>&, const DMatrix<double>&, lteUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,lteUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, lteUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,gtUnaryOp>(DMatrix<float>&, const DMatrix<float>&, gtUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,gtUnaryOp>(DMatrix<double>&, const DMatrix<double>&, gtUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,gtUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, gtUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,gteUnaryOp>(DMatrix<float>&, const DMatrix<float>&, gteUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,gteUnaryOp>(DMatrix<double>&, const DMatrix<double>&, gteUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,gteUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, gteUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,eqUnaryOp>(DMatrix<float>&, const DMatrix<float>&, eqUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,eqUnaryOp>(DMatrix<double>&, const DMatrix<double>&, eqUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,eqUnaryOp>(DMatrix<ulong>&, const DMatrix<ulong>&, eqUnaryOp<ulong>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,notAlmostEqUnaryOp>(DMatrix<float>&, const DMatrix<float>&, notAlmostEqUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,notAlmostEqUnaryOp>(DMatrix<double>&, const DMatrix<double>&, notAlmostEqUnaryOp<double>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<float,neqUnaryOp>(DMatrix<float>&, const DMatrix<float>&, neqUnaryOp<float>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,neqUnaryOp>(DMatrix<double>&, const DMatrix<double>&, neqUnaryOp<double>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<int, negateUnaryOp>(DMatrix<int>&, DMatrix<int> const&, negateUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, sigmoidUnaryOp>(DMatrix<int>&, DMatrix<int> const&, sigmoidUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, sigmoidGradientUnaryOp>(DMatrix<int>&, DMatrix<int> const&, sigmoidGradientUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, logUnaryOp>(DMatrix<int>&, DMatrix<int> const&, logUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, oneOverUnaryOp>(DMatrix<int>&, DMatrix<int> const&, oneOverUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, expUnaryOp>(DMatrix<int>&, DMatrix<int> const&, expUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, sqrtUnaryOp>(DMatrix<int>&, DMatrix<int> const&, sqrtUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, sqrUnaryOp>(DMatrix<int>&, DMatrix<int> const&, sqrUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, powUnaryOp>(DMatrix<int>&, DMatrix<int> const&, powUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, divSqrtUnaryOp>(DMatrix<int>&, DMatrix<int> const&, divSqrtUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, negateUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, negateUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, sigmoidUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, sigmoidUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, sigmoidGradientUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, sigmoidGradientUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, logUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, logUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, oneOverUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, oneOverUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, expUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, expUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, sqrtUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, sqrtUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, sqrUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, sqrUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, powUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, powUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, divSqrtUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, divSqrtUnaryOp<unsigned int>, int, ihipStream_t*);


template __host__ CUDART_DEVICE void unaryOpDmL<int, subFromUnaryOp>(DMatrix<int>&, DMatrix<int> const&, subFromUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, ltUnaryOp>(DMatrix<int>&, DMatrix<int> const&, ltUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, lteUnaryOp>(DMatrix<int>&, DMatrix<int> const&, lteUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, gtUnaryOp>(DMatrix<int>&, DMatrix<int> const&, gtUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, gteUnaryOp>(DMatrix<int>&, DMatrix<int> const&, gteUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, eqUnaryOp>(DMatrix<int>&, DMatrix<int> const&, eqUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, translationUnaryOp>(DMatrix<int>&, DMatrix<int> const&, translationUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<int, scaleUnaryOp>(DMatrix<int>&, DMatrix<int> const&, scaleUnaryOp<int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, subFromUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, subFromUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, ltUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, ltUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, lteUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, lteUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, gtUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, gtUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, gteUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, gteUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, eqUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, eqUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, translationUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, translationUnaryOp<unsigned int>, int, ihipStream_t*);
template __host__ CUDART_DEVICE void unaryOpDmL<unsigned int, scaleUnaryOp>(DMatrix<unsigned int>&, DMatrix<unsigned int> const&, scaleUnaryOp<unsigned int>, int, ihipStream_t*);

#else
template __host__ CUDART_DEVICE void unaryOpDmL<float,0>(DMatrix<float>&, const DMatrix<float>&, UnaryOpF<float,0>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,0>(DMatrix<double>&, const DMatrix<double>&, UnaryOpF<double,0>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<long,0>(DMatrix<long>&, const DMatrix<long>&, UnaryOpF<long,0>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,0>(DMatrix<ulong>&, const DMatrix<ulong>&, UnaryOpF<ulong,0>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<int,0>(DMatrix<int>&, const DMatrix<int>&, UnaryOpF<int,0>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<uint,0>(DMatrix<uint>&, const DMatrix<uint>&, UnaryOpF<uint,0>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,1>(DMatrix<float>&, const DMatrix<float>&, UnaryOpF<float,1>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,1>(DMatrix<double>&, const DMatrix<double>&, UnaryOpF<double,1>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<long,1>(DMatrix<long>&, const DMatrix<long>&, UnaryOpF<long,1>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,1>(DMatrix<ulong>&, const DMatrix<ulong>&, UnaryOpF<ulong,1>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<int,1>(DMatrix<int>&, const DMatrix<int>&, UnaryOpF<int,1>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<uint,1>(DMatrix<uint>&, const DMatrix<uint>&, UnaryOpF<uint,1>,int, ihipStream_t *);

template __host__ CUDART_DEVICE void unaryOpDmL<float,2>(DMatrix<float>&, const DMatrix<float>&, UnaryOpF<float,2>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<double,2>(DMatrix<double>&, const DMatrix<double>&, UnaryOpF<double,2>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<long,2>(DMatrix<long>&, const DMatrix<long>&, UnaryOpF<long,2>,int, ihipStream_t *);
template __host__ CUDART_DEVICE void unaryOpDmL<ulong,2>(DMatrix<ulong>&, const DMatrix<ulong>&, UnaryOpF<ulong,2>,int, ihipStream_t *);
#endif
