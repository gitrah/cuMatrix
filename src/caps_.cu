#include "hip/hip_runtime.h"
#include "caps.h"
#include "Kernels.h"
#include <hip/hip_runtime.h>
#include "MemMgr.h"

__device__ int d_MaxThreads = 512;
__device__ int d_MaxBlocks = 128;

__device__ ExecCaps** gd_devCaps = nullptr;
__constant__ int gd_devCount[MAX_GPUS];

 __host__ __device__ void getReductionExecContext(int &blocks, int &threads, long nP,int maxBlocks, int maxThreads) {
	int x = (nP + 1) / 2;
	if(x < 2) {
		 x=2;
	} else {
		--x;
		x |= x >> 1;
		x |= x >> 2;
		x |= x >> 4;
		x |= x >> 8;
		x |= x >> 16;
		++x;
	}

	threads =  nP == 2 ? 1 : (nP < (ulong) maxThreads * 2) ? x : maxThreads;
	blocks =  DIV_UP(nP, threads*2);

	blocks = MIN(maxBlocks, blocks);
	if(checkDebug(debugRedux))flprintf("np %d -> blocks %d of threads %d\n", nP, blocks, threads);
}

__host__ __device__ ExecCaps::~ExecCaps() {
	flprintf( "ExecCaps::~ExecCaps() this %p\n",this);
#ifndef __CUDA_ARCH__
	if(checkDebug(debugExec))b_util::dumpStack();
#endif
}

__host__ __device__ int ExecCaps::currDev() {
	int dev = 0;
#ifndef __CUDA_ARCH__
	cherr(hipGetDevice(&dev));
#endif
	return dev;
}
__host__ __device__ int ExecCaps::setDevice(const char* filename,const char* func, int line, int dev) {
	int orgDev = currDev();
#ifndef __CUDA_ARCH__
//	if(checkDebug(debugCheckValid))
		if(orgDev != dev) {
			checkCudaErrors(hipSetDevice(dev));
			printf( "%s:%d %s changing device to %d from %d\n", filename, line, func, dev, orgDev);
		}
#endif
	return orgDev;
}
__host__ __device__ int ExecCaps::visitDevice(const char* filename,const char* func, int line, int dev) {
	int orgDev = currDev();
#ifndef __CUDA_ARCH__
//	if(checkDebug(debugCheckValid))
		if(orgDev != dev) {
			checkCudaErrors(hipSetDevice(dev));
			if(checkDebug(debugExec))printf( "%s:%d %s visiting (temporarily) device %d from %d\n", filename, line, func, dev, orgDev);
		}
#endif
	return orgDev;
}

__host__ __device__ int ExecCaps::restoreDevice(const char* filename,const char* func, int line, int dev) {
	int orgDev = currDev();
#ifndef __CUDA_ARCH__
//	if(checkDebug(debugCheckValid))
		if(orgDev != dev) {
			cherr(hipSetDevice(dev));
			if(checkDebug(debugExec))printf( "%s:%d %s restoring device %d from %d\n", filename, line, func, dev, orgDev);
		}
#endif
	return orgDev;
}


__global__ void freeDevSideDevCaps() {
	//flprintf( "freeDevSideDevCaps freeing gd_devCaps %p gd_devCount %d \n", gd_devCaps, gd_devCount);
	prlocf("freeDevSideDevCaps\n");
	FirstThread {
		for(int i = 0; i < ExecCaps::countGpus(); i++) {
			flprintf("freeDevSideDevCaps deleting gd_devCaps[%d] %p\n", i, gd_devCaps[i]);
			delete gd_devCaps[i];
		}
		flprintf( "freeinbg gd_devCaps %p\n", gd_devCaps);
		free(gd_devCaps);
	}
}

void ExecCaps::freeDevCaps() {
	flprintf( "ExecCaps::freeDevCaps freeDevCaps enter this %d\n",0);
	for(int i = 0; i < ExecCaps::countGpus(); i++) {
		flprintf("ExecCaps::freeDevCaps deleting g_devCaps[device = %d] %p\n",i, g_devCaps[i]);
		delete g_devCaps[i];
	}
	flprintf( "ExecCaps::freeDevCaps freeing g_devCaps %p\n",g_devCaps);
	free(g_devCaps);
	//freeDevSideDevCaps<<<1,1>>>();
}


__host__ __device__ hipError_t ExecCaps::currCaps(ExecCaps** caps, int dev) {

#ifndef __CUDA_ARCH__
	if(dev < ExecCaps::countGpus()) {
		*caps = g_devCaps[dev];
		return hipSuccess;
	}
	return hipErrorUnknown;
#else
	if(true) {
	//s	setLastError(notImplementedEx);
		return hipErrorAssert;
	}
#endif

}

__host__ __device__ ExecCaps* ExecCaps::currCaps(int dev ) {
#ifndef __CUDA_ARCH__
	if(checkDebug(debugExec))flprintf( "in ExecCaps::currCaps ExecCaps::countGpus() %d\n",ExecCaps::countGpus());
	if(dev < ExecCaps::countGpus()) {
		return g_devCaps[dev];
	}
#else
	if(checkDebug(debugExec))flprintf( "in [D}ExecCaps::currCaps gd_devCount %d\n",gd_devCount);
//	setLastError(notImplementedEx);
/*	if(dev < ExecCaps::countGpus()) {*/
		ExecCaps* pCaps = gd_devCaps[dev];
		flprintf( "pCaps %p \n ",pCaps);
		flprintf( "gd_devCaps[%d] %p \n ",dev, pCaps);

		return pCaps;
/*	}*/
#endif
	return nullptr;
}


__host__ __device__ void ExecCaps::printMaxDims(const char* msg) {
	printf("%s for dev %d maxGrid(%u,%u,%u)", msg, devNumber, maxGrid.x,maxGrid.y,maxGrid.z);
	printf("maxBlock(%u,%u,%u)", maxBlock.x,maxBlock.y,maxBlock.z);
}


/*
__host__ __device__  hipError_t ExecCaps::currStream(hipStream_t* stream,int dev) {
	ExecCaps* currCaps = null;
	hipError_t res = ExecCaps::currCaps(&currCaps,dev) ;
	if(res != hipSuccess) {
		cherr(res);
		return res;
	}
	if(currCaps) {
		if(checkDebug(debugStream))flprintf( "currStream(dev = %d) -> %p\n", dev, currCaps->stream);
		*stream = currCaps->stream;
		return hipSuccess;
	}
	*stream = null;
	return hipErrorUnknown;
}
*/
/*

__host__ __device__ hipStream_t ExecCaps::currStream(int dev) {
	hipStream_t currStream;
	hipError_t res = ExecCaps::currStream(&currStream, dev);
	if(res != hipSuccess) {
		if(checkDebug(debugStream))flprintf( "currStream error for dev %d\n", dev);
	}
	cherr(res);
	return currStream;
}

*/
__host__ __device__ const char *__cudaGetErrorEnum(hipError_t error)
{
    switch (error)
    {
        case hipSuccess:
            return "hipSuccess";

        case hipErrorMissingConfiguration:
            return "hipErrorMissingConfiguration";

        case hipErrorOutOfMemory:
            return "hipErrorOutOfMemory";

        case hipErrorNotInitialized:
            return "hipErrorNotInitialized";

        case hipErrorLaunchFailure:
            return "hipErrorLaunchFailure";

        case hipErrorPriorLaunchFailure:
            return "hipErrorPriorLaunchFailure";

        case hipErrorLaunchTimeOut:
            return "hipErrorLaunchTimeOut";

        case hipErrorLaunchOutOfResources:
            return "hipErrorLaunchOutOfResources";

        case hipErrorInvalidDeviceFunction:
            return "hipErrorInvalidDeviceFunction";

        case hipErrorInvalidConfiguration:
            return "hipErrorInvalidConfiguration";

        case hipErrorInvalidDevice:
            return "hipErrorInvalidDevice";

        case hipErrorInvalidValue:
            return "hipErrorInvalidValue";

        case hipErrorInvalidPitchValue:
            return "hipErrorInvalidPitchValue";

        case hipErrorInvalidSymbol:
            return "hipErrorInvalidSymbol";

        case hipErrorMapFailed:
            return "hipErrorMapFailed";

        case hipErrorUnmapFailed:
            return "hipErrorUnmapFailed";

        case cudaErrorInvalidHostPointer:
            return "cudaErrorInvalidHostPointer";

        case hipErrorInvalidDevicePointer:
            return "hipErrorInvalidDevicePointer";

        case hipErrorInvalidTexture:
            return "hipErrorInvalidTexture";

        case cudaErrorInvalidTextureBinding:
            return "cudaErrorInvalidTextureBinding";

        case hipErrorInvalidChannelDescriptor:
            return "hipErrorInvalidChannelDescriptor";

        case hipErrorInvalidMemcpyDirection:
            return "hipErrorInvalidMemcpyDirection";

        case cudaErrorAddressOfConstant:
            return "cudaErrorAddressOfConstant";

        case cudaErrorTextureFetchFailed:
            return "cudaErrorTextureFetchFailed";

        case cudaErrorTextureNotBound:
            return "cudaErrorTextureNotBound";

        case cudaErrorSynchronizationError:
            return "cudaErrorSynchronizationError";

        case cudaErrorInvalidFilterSetting:
            return "cudaErrorInvalidFilterSetting";

        case cudaErrorInvalidNormSetting:
            return "cudaErrorInvalidNormSetting";

        case cudaErrorMixedDeviceExecution:
            return "cudaErrorMixedDeviceExecution";

        case hipErrorDeinitialized:
            return "hipErrorDeinitialized";

        case hipErrorUnknown:
            return "hipErrorUnknown";

        case cudaErrorNotYetImplemented:
            return "cudaErrorNotYetImplemented";

        case cudaErrorMemoryValueTooLarge:
            return "cudaErrorMemoryValueTooLarge";

        case hipErrorInvalidHandle:
            return "hipErrorInvalidHandle";

        case hipErrorNotReady:
            return "hipErrorNotReady";

        case hipErrorInsufficientDriver:
            return "hipErrorInsufficientDriver";

        case hipErrorSetOnActiveProcess:
            return "hipErrorSetOnActiveProcess";

        case cudaErrorInvalidSurface:
            return "cudaErrorInvalidSurface";

        case hipErrorNoDevice:
            return "hipErrorNoDevice";

        case hipErrorECCNotCorrectable:
            return "hipErrorECCNotCorrectable";

        case hipErrorSharedObjectSymbolNotFound:
            return "hipErrorSharedObjectSymbolNotFound";

        case hipErrorSharedObjectInitFailed:
            return "hipErrorSharedObjectInitFailed";

        case hipErrorUnsupportedLimit:
            return "hipErrorUnsupportedLimit";

        case cudaErrorDuplicateVariableName:
            return "cudaErrorDuplicateVariableName";

        case cudaErrorDuplicateTextureName:
            return "cudaErrorDuplicateTextureName";

        case cudaErrorDuplicateSurfaceName:
            return "cudaErrorDuplicateSurfaceName";

        case cudaErrorDevicesUnavailable:
            return "cudaErrorDevicesUnavailable";

        case hipErrorInvalidImage:
            return "hipErrorInvalidImage";

        case hipErrorNoBinaryForGpu:
            return "hipErrorNoBinaryForGpu";

        case cudaErrorIncompatibleDriverContext:
            return "cudaErrorIncompatibleDriverContext";

        case hipErrorPeerAccessAlreadyEnabled:
            return "hipErrorPeerAccessAlreadyEnabled";

        case hipErrorPeerAccessNotEnabled:
            return "hipErrorPeerAccessNotEnabled";

        case hipErrorContextAlreadyInUse:
            return "hipErrorContextAlreadyInUse";

        case hipErrorProfilerDisabled:
            return "hipErrorProfilerDisabled";

        case hipErrorProfilerNotInitialized:
            return "hipErrorProfilerNotInitialized";

        case hipErrorProfilerAlreadyStarted:
            return "hipErrorProfilerAlreadyStarted";

        case hipErrorProfilerAlreadyStopped:
            return "hipErrorProfilerAlreadyStopped";

#if __CUDA_API_VERSION >= 0x4000

        case hipErrorAssert:
            return "hipErrorAssert";

        case cudaErrorTooManyPeers:
            return "cudaErrorTooManyPeers";

        case hipErrorHostMemoryAlreadyRegistered:
            return "hipErrorHostMemoryAlreadyRegistered";

        case hipErrorHostMemoryNotRegistered:
            return "hipErrorHostMemoryNotRegistered";
#endif

//#if __CUDA_API_VERSION >= 6000
        case hipErrorOperatingSystem:
            return "hipErrorOperatingSystem";
        case hipErrorPeerAccessUnsupported:
            return "hipErrorPeerAccessUnsupported";
        case cudaErrorLaunchMaxDepthExceeded:
            return "cudaErrorLaunchMaxDepthExceeded";
        case cudaErrorLaunchFileScopedTex:
            return "cudaErrorLaunchFileScopedTex";
        case cudaErrorLaunchFileScopedSurf:
            return "cudaErrorLaunchFileScopedSurf";
        case cudaErrorSyncDepthExceeded:
            return "cudaErrorSyncDepthExceeded";
        case cudaErrorLaunchPendingCountExceeded:
            return "cudaErrorLaunchPendingCountExceeded";
        case cudaErrorNotPermitted:
            return "cudaErrorNotPermitted";
        case hipErrorNotSupported:
            return "hipErrorNotSupported";
        case cudaErrorHardwareStackError:
            return "cudaErrorHardwareStackError";
        case cudaErrorIllegalInstruction:
            return "cudaErrorIllegalInstruction";
        case cudaErrorMisalignedAddress:
            return "cudaErrorMisalignedAddress";
        case cudaErrorInvalidAddressSpace:
            return "cudaErrorInvalidAddressSpace";
        case cudaErrorInvalidPc:
            return "cudaErrorInvalidPc";
        case hipErrorIllegalAddress:
            return "hipErrorIllegalAddress";
//#endif
        case cudaErrorStartupFailure:
            return "cudaErrorStartupFailure";
        case cudaErrorApiFailureBase:
            return "cudaErrorApiFailureBase";
    }

    return "<unknown>";
}

__host__ __device__ const char *__cublasGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
    case HIPBLAS_STATUS_SUCCESS:
    	return "HIPBLAS_STATUS_SUCCESS =0";
    case   HIPBLAS_STATUS_NOT_INITIALIZED:
    	return "HIPBLAS_STATUS_NOT_INITIALIZED =1";
    case HIPBLAS_STATUS_ALLOC_FAILED:
    	return "HIPBLAS_STATUS_ALLOC_FAILED    =3";
    case HIPBLAS_STATUS_INVALID_VALUE:
    	return "HIPBLAS_STATUS_INVALID_VALUE   =7";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
    	return "HIPBLAS_STATUS_ARCH_MISMATCH   =8";
    case HIPBLAS_STATUS_MAPPING_ERROR:
    	return "HIPBLAS_STATUS_MAPPING_ERROR   =11";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
    	return "HIPBLAS_STATUS_EXECUTION_FAILED=13";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
    	return "HIPBLAS_STATUS_INTERNAL_ERROR  =14";
    case HIPBLAS_STATUS_NOT_SUPPORTED:
    	return "HIPBLAS_STATUS_NOT_SUPPORTED   =15";
    case HIPBLAS_STATUS_UNKNOWN:
    	return "HIPBLAS_STATUS_UNKNOWN   =16";
    default:
    	return "UNKNOWN";
    }


}

__host__ __device__ int ExecCaps::maxThreads() {
#ifndef __CUDA_ARCH__
		return MaxThreads;
#else
		return d_MaxThreads;
#endif
}
__host__ __device__ int ExecCaps::maxBlocks() {
#ifndef __CUDA_ARCH__
		return MaxBlocks;
#else
		return d_MaxBlocks;
#endif
}

/*
 * survey all gpus and find the smallest 'reasonable' max buffer (as headroom fraction of total)
 */
__host__ __device__ size_t ExecCaps::minMaxReasonable(int gpuMask, float headroom) {
	int devCnt;
	cherr(hipPeekAtLastError());
	checkCudaError(hipGetDeviceCount(&devCnt));
	size_t minMax = 0, currMax = 0;

	for(int i = 0; i < devCnt; i++) {
		if(gpuMask & (1 << i)) {
			currMax = ExecCaps::currCaps(i)->maxReasonable(headroom);
			if(checkDebug(debugMem))flprintf("device %d has total %lu maxreas %lu (at %2.2f head)\n", i,  ExecCaps::currCaps(i)->deviceProp.totalGlobalMem, currMax, headroom);
			minMax = minMax == 0 ? currMax : MIN(minMax, currMax);
		}
	}
	return minMax;
}

__global__ void createCapsPPtr(int devCount) {
	FirstThread {
		gd_devCaps = (ExecCaps**) malloc(devCount* sizeof(ExecCaps*));
		flprintf("createCapsPPtr created %p (%d bytes)\n",gd_devCaps, sizeof(ExecCaps*));
	}
}

__global__ void addCaps(int dev, ExecCaps caps) {
	FirstThread {
		ExecCaps* pcaps = new ExecCaps(caps);
		gd_devCaps[dev] = pcaps;
		memcpy(pcaps, &caps, sizeof(ExecCaps));
		flprintf("addCaps created gd_devCaps[%d] %p (%d bytes)\n",dev, pcaps, sizeof(ExecCaps));
	}
}

hipError_t ExecCaps::addDevice(int dev) {
	ExecCaps_setDevice(dev);
//	checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gd_devCount), (void*) &ExecCaps::countGpus(), sizeof(int)));
 //	hipDeviceGet(&device,dev);

	createCapsPPtr<<<1,1>>>(ExecCaps::countGpus());
	addCaps<<<1,1>>>(dev, *g_devCaps[dev]);
	return hipSuccess;
}

void ExecCaps::initDevCaps() {
	//flprintf("%s enter\n","ExecCaps::initDevCaps");
	if(checkDebug(debugVerbose))prlocf("enter\n");
	//ExecCaps* g_devCaps[];
	g_devCaps = (ExecCaps**) malloc(ExecCaps::countGpus()* sizeof(ExecCaps*));
	if(checkDebug(debugVerbose))flprintf("gpuCount %d, malloced-> %d\n",ExecCaps::countGpus(),ExecCaps::countGpus()* sizeof(ExecCaps*) );
	//outln("gpuCount " << ExecCaps::countGpus() << " mallocd " <<(ExecCaps::countGpus()* sizeof(ExecCaps*)));
	for(int i = 0; i < ExecCaps::countGpus(); i++) {
		ExecCaps* cap = new ExecCaps();
		if(checkDebug(debugVerbose))flprintf("created cap %p\n", cap);
		g_devCaps[i] = cap;
		ExecCaps::getExecCaps(*cap, i);
		if(checkDebug(debugVerbose))outln("adding " << i << "\n" << cap->toString() << "\n");
		ExecCaps::addDevice(i);
	}
}

__host__ void ExecCaps::allGpuMem(size_t* free, size_t* total) {
	if(checkDebug(debugVerbose))outln("allGpuMem free " << free <<  ", total " << total);
    int orgDev = ExecCaps::currDev();
    if(checkDebug(debugVerbose))outln("orgDev " << orgDev);
    *free=0;
    *total=0;
    size_t lFree = 0, lTotal = 0;
	int devCnt;
	char buff[20];
	cherr(hipGetDeviceCount(&devCnt));
	if(checkDebug(debugVerbose))outln("devCnt " << devCnt);
//cout << "  ";
	float* dtest = nullptr;

	b_util::dumpStack();
	for(int i = devCnt -1; i >-1;i--) {
		ExecCaps_visitDevice(i);
		if(checkDebug(debugVerbose))outln("ExecCaps_setDevice  " << i );
		cherr(hipMalloc(&dtest, 10 * sizeof(float)));
		if(checkDebug(debugVerbose))outln("hipMalloc  dtest " << dtest << "...checking valid") ;
		MemMgr<float>::checkValid(dtest, "ExecCaps::allGpuMem dtest ");
		if(checkDebug(debugVerbose))outln("ExecCaps_setDevice  " << i );
		cherr(hipMemGetInfo(&lFree, &lTotal));
		*free += lFree;
		*total += lTotal;
		sprintf(buff, " (%.2f%% used)", 100 * (1 - lFree * 1. / lTotal));
		if(checkDebug(debugMem)) cout << "[" << i <<":  " <<  b_util::expNotation(lFree) << " free /" << b_util::expNotation(lTotal) << buff<< "] ";
	}
	//cout << endl;
	ExecCaps_setDevice(orgDev);
}
__host__ __device__ DevStream::DevStream(int device) :  device(device) {
	int orgDev = ExecCaps::currDev();
	if(orgDev != device)
		ExecCaps_visitDevice(device);

#ifndef __CUDA__ARCH__
	cherr(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
	flprintf("on gpieux %d, created DevStream(%d,%p)\n",  ExecCaps::currDev(), device, stream);
	if(orgDev != device)
		ExecCaps_setDevice(orgDev);
#else
	stream = 0;
#endif
}

__host__ __device__ hipError_t  DevStream::sync() {
	int orgDev = ExecCaps::currDev();
	if(orgDev != device)
		ExecCaps_visitDevice(device);
	flprintf("on gpieux  %d syncing DevStream(%d,%p)\n",  ExecCaps::currDev(), device, stream);
#ifndef __CUDA_ARCH__
	hipError_t res =  hipStreamSynchronize(stream);
#else
	hipError_t res =  hipDeviceSynchronize();
#endif
	flprintf("sanched  -> %s\n", __cudaGetErrorEnum(res) );
	if(orgDev != device)
		ExecCaps_setDevice(orgDev);

	return res;
}
__host__ __device__ hipError_t DevStream::destroy() {
	return hipStreamDestroy(stream);
}

__host__ __device__ DevStream::~DevStream() {
	cherr(destroy());
}
