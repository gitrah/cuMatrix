#include "hip/hip_runtime.h"
/*
 * CuMatrixTranspose.cu
 *
 */
#include "CuMatrix.h"
#include "util.h"
#include "debug.h"
#include "caps.h"
#include "MatrixExceptions.h"

int blockH = TX_BLOCK_SIZE/4;

// tiles must be square
template<typename T> __global__ void transposeNaive(T* tElements,
		const T* sElements, int width, int height, int spitch, int tpitch) {
	int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int yIndex = blockIdx.y * blockDim.x + threadIdx.y; // not blockDim.y, which we assume a factor of blockDim.x

	int index_in = xIndex + spitch * yIndex;
	int index_out = yIndex + tpitch * xIndex;
	// threads must iterate normal to the cache line,
	// which doesn't happen in this write to t
	for (int i = 0; i < blockDim.x; i += blockDim.y)
		if (xIndex < width && yIndex + i < height)
			tElements[index_out + i] = sElements[index_in + i * spitch];
}
template void __global__ transposeNaive<float>(float*,const float*,int,int,int,int);
template void __global__ transposeNaive<double>(double*,const double*,int,int,int,int);
template void __global__ transposeNaive<ulong>(ulong*,const ulong*,int,int,int,int);

template <typename T> __global__ void transposeSubTile(T* tElements, const T* sElements, int width, int height, int spitch, int tpitch)
{
	T* tile = SharedMemory<T>();
    uint xIndex = threadIdx.x;
    uint yIndex = threadIdx.y;
	uint vmemIdx = yIndex * spitch + xIndex;
	uint txIdx = xIndex * tpitch + yIndex;

	if(xIndex < width && yIndex < height) {
		tile[txIdx] = sElements[vmemIdx];
	}
	__syncthreads();
	if(xIndex < width && yIndex < height) {
		tElements[vmemIdx] = tile[vmemIdx];
	}
}

template <typename T> __global__ void transposeCoalesced(T* tElements, const T* sElements, int width, int height, int spitch, int tpitch)
{
	T* tile = SharedMemory<T>();

    uint xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    uint yIndex = blockIdx.y * blockDim.x + threadIdx.y;
    uint index_in = xIndex + yIndex * spitch;
    uint tileIdxOut = threadIdx.x * blockDim.x + threadIdx.y;
    uint tileIdxIn = threadIdx.y * blockDim.x + threadIdx.x;

    for(int i = 0; i < blockDim.x; i += blockDim.y)
    	if(xIndex < width && yIndex + i < height )
    		tile[tileIdxIn + i * blockDim.x] = sElements[index_in + i * spitch];

    __syncthreads();

    xIndex = blockIdx.y * blockDim.x + threadIdx.x;
    yIndex = blockIdx.x * blockDim.x + threadIdx.y; // not blockDim.y
    int index_out = xIndex + yIndex * tpitch;
    for(int i = 0; i < blockDim.x; i += blockDim.y)
    	if(xIndex < height && yIndex + i < width)
    		tElements[index_out + i * tpitch] = tile[tileIdxOut + i];
}
template void __global__ transposeCoalesced<float>( float*,const float*,int,int,int,int);
template void __global__ transposeCoalesced<double>(double*,const double*,int,int,int,int);
template void __global__ transposeCoalesced<ulong>(ulong*,const ulong*,int,int,int,int);

template <typename T>
__global__ void transposeNoBankConflicts(T* tElements, const T* sElements, int width, int height)
{
	T* tile = SharedMemory<T>();

    uint xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    uint yIndex = blockIdx.y * blockDim.x + threadIdx.y;
    uint index_in = xIndex + yIndex*width;
    uint tileIdxIn = threadIdx.y * (blockDim.x + 1) + threadIdx.x;
    uint tileIdxOut = threadIdx.x * (blockDim.x + 1) + threadIdx.y;

    for(int i = 0; i < blockDim.x; i += blockDim.y)
    	if(xIndex < width && yIndex + i < height )
    		tile[tileIdxIn + i* ( blockDim.x +1)] = sElements[index_in + i * width];
    __syncthreads();
    xIndex = blockIdx.y * blockDim.x + threadIdx.x;
    yIndex = blockIdx.x * blockDim.x + threadIdx.y;
    int index_out = xIndex + (yIndex)*height;
    for(int i = 0; i < blockDim.x; i += blockDim.y)
    	if(xIndex < height && yIndex + i < width)
    		tElements[index_out + i * height] = tile[tileIdxOut + i];

}

template <typename T>
__global__ void transposeNoBankConflictsPitch(T* tElements, const T* sElements, int width, int height, int spitch, int tpitch)
{
	T* tile = SharedMemory<T>();

    uint xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    uint yIndex = blockIdx.y * blockDim.x + threadIdx.y;
    uint index_in = xIndex + yIndex*spitch;
    uint tileIdxIn = threadIdx.y * (blockDim.x + 1) + threadIdx.x;
    uint tileIdxOut = threadIdx.x * (blockDim.x + 1) + threadIdx.y;

    for(int i = 0; i < blockDim.x; i += blockDim.y)
    	if(xIndex < width && yIndex + i < height )
    		tile[tileIdxIn + i* ( blockDim.x +1)] = sElements[index_in + i * spitch];
    __syncthreads();
    xIndex = blockIdx.y * blockDim.x + threadIdx.x;
    yIndex = blockIdx.x * blockDim.x + threadIdx.y;
    int index_out = xIndex + (yIndex)*tpitch;
    for(int i = 0; i < blockDim.x; i += blockDim.y)
    	if(xIndex < height && yIndex + i < width)
    		tElements[index_out + i * tpitch] = tile[tileIdxOut + i];

}

template <typename T> __global__ void transposeDiagonalKernel( T* tElements, const T* sElements, int width, int height)
{
	T* tile = SharedMemory<T>();

	int blockIdx_x, blockIdx_y;

	if (width == height) {
		blockIdx_y = blockIdx.x;
		blockIdx_x = (blockIdx.x + blockIdx.y) % gridDim.x;
	} else {
        int bid = blockIdx.x + gridDim.x*blockIdx.y;
        blockIdx_y = bid % gridDim.y;
        blockIdx_x = ((bid/gridDim.y)+blockIdx_y)%gridDim.x;
    }
    uint xIndex = blockIdx_x * blockDim.x + threadIdx.x;
    uint yIndex = blockIdx_y * blockDim.x + threadIdx.y;
    uint index_in = xIndex + (yIndex)*width;
    uint tileIdx = threadIdx.y * (blockDim.x + 1)+ threadIdx.x;
    uint tileIdxOut = threadIdx.x * (blockDim.x + 1) + threadIdx.y;

    for(int i = 0; i < blockDim.x; i += blockDim.y)
    	if(xIndex < width && yIndex + i < height )
    		tile[tileIdx + i * (blockDim.x + 1)] = sElements[index_in+i*width];

    __syncthreads();
    xIndex = blockIdx_y * blockDim.x + threadIdx.x;
    yIndex = blockIdx_x * blockDim.x + threadIdx.y;
    int index_out = xIndex + (yIndex)*height;
    for(int i = 0; i < blockDim.x; i += blockDim.y)
    	if(xIndex < height && yIndex + i < width)
    		tElements[index_out + i * height] = tile[tileIdxOut + i];

}
template <typename T> __global__ void transposeDiagonalPitchKernel( T* tElements, const T* sElements, int width, int height, int spitch, int tpitch)
{
	T* tile = SharedMemory<T>();

	int blockIdx_x, blockIdx_y;

	if (width == height) {
		blockIdx_y = blockIdx.x;
		blockIdx_x = (blockIdx.x + blockIdx.y) % gridDim.x;
	} else {
        int bid = blockIdx.x + gridDim.x*blockIdx.y;
        blockIdx_y = bid % gridDim.y;
        blockIdx_x = ((bid/gridDim.y)+blockIdx_y)%gridDim.x;
    }
    uint xIndex = blockIdx_x * blockDim.x + threadIdx.x;
    uint yIndex = blockIdx_y * blockDim.x + threadIdx.y;
    uint index_in = xIndex + (yIndex)*spitch;
    uint tileIdx = threadIdx.y * (blockDim.x + 1)+ threadIdx.x;
    uint tileIdxOut = threadIdx.x * (blockDim.x + 1) + threadIdx.y;

    for(int i = 0; i < blockDim.x; i += blockDim.y)
    	if(xIndex < width && yIndex + i < height )
    		tile[tileIdx + i * (blockDim.x + 1)] = sElements[index_in+i*spitch];

    __syncthreads();
    xIndex = blockIdx_y * blockDim.x + threadIdx.x;
    yIndex = blockIdx_x * blockDim.x + threadIdx.y;
    int index_out = xIndex + (yIndex)*tpitch;
    for(int i = 0; i < blockDim.x; i += blockDim.y)
    	if(xIndex < height && yIndex + i < width)
    		tElements[index_out + i * tpitch] = tile[tileIdxOut + i];

}
template <typename T> __host__ CUDART_DEVICE void CuMatrix<T>::transposeKernelPtrL(DMatrix<T>& t,void (*kernel)( T*, const T*, int,int), const DMatrix<T>& s , hipStream_t stream )  {
	ulong len = s.m * s.n;
	assert( len == t.m * t.n );
	int blockW = TX_BLOCK_SIZE;
	dim3 block;
	defaultBlock(block);
    dim3 grid(DIV_UP(s.n, blockW), DIV_UP(s.m, blockW));

    void (*txNmbcPtr)( T*,const T*,int,int);
    txNmbcPtr=&transposeNoBankConflicts;
    void (*txDiagPtr)(T*,const T*,int,int);
    txDiagPtr=&transposeDiagonalKernel;
	int tileWidth = blockW;
	if(kernel == txNmbcPtr || kernel == txDiagPtr) {
		tileWidth++;
	}
	int smem = TX_BLOCK_SIZE * (tileWidth)* sizeof(T);
	kernel<<<grid, block, smem, stream>>>(t.elements, s.elements, s.n, s.m );
	//outln("tx with grid " << b_util::pd3(grid).c_str() << " of block " << b_util::pd3(block).c_str() << " smem " << smem);
}
template <typename T> __host__ CUDART_DEVICE void CuMatrix<T>::transposePitchKernelPtrL(DMatrix<T>& t,void (*kernel)( T*, const T*, int, int,int,int), const DMatrix<T>& s , hipStream_t stream )  {
	ulong len = s.m * s.n;
	assert( len == t.m * t.n );
	int blockW = TX_BLOCK_SIZE;
	dim3 block;
	defaultBlock(block);
    dim3 grid(DIV_UP(s.n, blockW), DIV_UP(s.m, blockW));

    static void (*txNmbcPtr)( T*,const T*,int,int,int,int);
    txNmbcPtr=&transposeNoBankConflictsPitch;
    static void (*txDiagPtr)(T*,const T*,int,int,int,int);
    txDiagPtr=&transposeDiagonalPitchKernel;
	int tileWidth = blockW;
	if(kernel == txNmbcPtr || kernel == txDiagPtr) {
		tileWidth++;
	}
	int smem = TX_BLOCK_SIZE * (tileWidth)* sizeof(T);
	kernel<<<grid, block, smem, stream>>>(t.elements, s.elements, s.n, s.m, s.p, t.p);
	//outln("tx with grid " << b_util::pd3(grid).c_str() << " of block " << b_util::pd3(block).c_str() << " smem " << smem);
}

template <typename T> __host__ CUDART_DEVICE void CuMatrix<T>::transposeL( DMatrix<T>& t, const DMatrix<T>& s, hipStream_t stream)  {\

    void (*txNmbcPtr)( T*,const T*,int,int,int,int);
    txNmbcPtr=&transposeDiagonalPitchKernel;

#ifndef __CUDA_ARCH__
    int tdev =  b_util::getDevice((void*)t.elements);
    int sdev =  b_util::getDevice((void*)s.elements);
    if(checkDebug(debugTxp))flprintf("trg elems %p (dev %d) src %p (dev %d)\n", t.elements,tdev, s.elements, sdev);
    assert( tdev == sdev );
#endif

    //txNmbcPtr=&transposeNoBankConflicts;
    transposePitchKernelPtrL(t, txNmbcPtr,s);
	cherr(hipDeviceSynchronize());
}

template<typename T> CuMatrix<T> CuMatrix<T>::transposeKernelPtr(
		void (*kernel)(T* sElements, const T* tElements, int width,
				int height)) {

	if (vectorQ()) {
		if (checkDebug(debugTxp))
			prlocf("degenerate tx");
		CuMatrix<T> ret = copy(true);
		ret.m = n;
		ret.n = m;
		ret.p = m;
		ret._tileP = m;
		return ret;
	}
	assert(tiler.tileSize >= tiler.m_size);
	CuMatrix<T> ret(n, m, true, true);
#ifndef __CUDA_ARCH__
	if(checkDebug(debugTxp))outln("tx on dev " << ExecCaps::currDev() << " from " << this->toShortString() << " to " << ret.toShortString() );
#endif
	DMatrix<T> retD, d_A;
	tile0(d_A, lastMod == mod_host);
	ret.tile0(retD, false);
	transposeKernelPtrL(retD, kernel, d_A);
	ret.invalidateHost();
	return ret;
}

template<typename T> CuMatrix<T> CuMatrix<T>::transposePitchKernelPtr(
		void (*kernel)(T*, const T*, int, int, int, int)) {

	if (vectorQ()) {
		if (checkDebug(debugTxp))
			prlocf("degenerate tx");
		CuMatrix<T> ret = copy(true);
		ret.m = n;
		ret.n = m;
		ret.p = m;
		return ret;
	}
	assert(tiler.tileSize >= tiler.m_size);
	CuMatrix<T> ret(n, m, true, true);
#ifndef __CUDA_ARCH__
	if(checkDebug(debugTxp))outln("tx on dev " << ExecCaps::currDev() << " from " << this->toShortString() << " to " << ret.toShortString() );
#endif
	DMatrix<T> retD, d_A;
	tile0(d_A, lastMod == mod_host);
	ret.tile0(retD, false);
	transposePitchKernelPtrL(retD, kernel, d_A);
	ret.invalidateHost();
	return ret;
}


template<typename T>  __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::transpose(hipStream_t stream ) const {
	if(scalarQ()) {
		return *this;
	}
	if(vectorQ() && n == p) {
		if(checkDebug(debugTxp)) prlocf("transpose() on nonaliased vector");
		CuMatrix<T> ret = copy(true);
		ret.m = n;
		ret.n = m;
		ret.p = m;
		ret.tiler.m_m = n;
		ret.tiler.m_n = m;
		ret.tiler.m_p = m;
		ret._tileP = m;
/*
		ret.ownsDBuffers = ownsDBuffers;
		ret.ownsHBuffers = ownsHBuffers;
		ret.elements = elements;
		ret.tiler = tiler;
		ret.tiler.m_m = n;
		ret.tiler.m_n = m;
		ret.lastMod = lastMod;
		ret.size = size;
		if(ret.tiler.hasDmemQ()) ret.getMgr().addTiles(&(ret.tiler));
		if(ret.elements) ret.getMgr().addHost(ret);
		if(checkDebug(debugTxp)) outln("spoofing transpose for column/row matrix " << toShortString());
*/
		return ret;
	}
	if(tiler.tileSize < tiler.m_size) {
		return transposeXr();
	}
	CuMatrix<T> ret(n,m, true,true);
	DMatrix<T> retD, d_A;
	tile0(d_A, lastMod == mod_host);
	ret.tile0(retD, false);
	transposeL(retD, d_A, stream );
	ret.invalidateHost();

	return ret;
}


template<typename T>  __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::transposeXr(hipStream_t stream) const {
	CuMatrix<T> ret(n,m, true,true);
	/*
	 *  tile of source starts at 0,0 and moves right
	 *  tiler of target starts at 0,0  and moves down
	 *  each subs source starts at lr corner of last start tile and moves right
	 *
	 tile2D(DMatrix<T>& dm,
            uint& roff, uint& coff,
            uint& tileM, uint& tileN,
            int rowTileIdx, int colTileIdx,
            int rowTileCount, int colTileCount,
            bool copy = true, int lastGpu =-1, hipStream_t stream = 0)

	 */


	DMatrix<T> d_A, d_B;

	int maxTileD = (int) (sqrtf( (float) tiler.tileSize) * MIN(m,n)/MAX(m,n));
	int tileD = b_util::prevPowerOf2(maxTileD);
	tileD = MIN(m, MIN(n, tileD));
	int colSteps = DIV_UP(n,tileD);
	int rowSteps = DIV_UP(m, tileD);

	const Tiler<T>* btiler =&(ret.tiler);

	int aroff = 0,acoff = 0;
	int lastGpu = -1;
	int gpuCount = tiler.countGpus();
	int orgDevice = ExecCaps::currDev();
    int rowTileIdx, colTileIdx;
    int rowTileCount, colTileCount;

	hipStream_t* streams = nullptr;
	lastGpu = tiler.nextGpu(lastGpu);
	int tileM = _tileM, tileN = _tileN, tileP = _tileP;
	if(gpuCount > 1) {
		assert(!stream);
		hipStream_t* streams = (hipStream_t* ) malloc(gpuCount * sizeof(hipStream_t));
		for(int i =0 ; i < gpuCount; i++) {
			lastGpu = tiler.nextGpu(lastGpu);
			ExecCaps_setDevice(lastGpu);
			cherr(hipStreamCreateWithFlags(&streams[i],hipStreamNonBlocking));
		}
	}

	int coliStart = 0;
    for(int rowi = 0; rowi < rowSteps; rowi++) {
    	for(int coli = coliStart; coli < colSteps; coli++) {
    		tiler.tile2D(d_A, aroff, acoff, tileM, tileN, tileP, rowSteps, colSteps, rowi, coli, true,  lastGpu, gpuCount > 1 ? streams[coli] : stream);
    		btiler->tile2D(d_B, acoff, aroff, tileM, tileN, tileP, rowSteps, colSteps, rowi, coli, false, lastGpu, gpuCount > 1 ? streams[coli] : stream);
    		transposeL(d_A,d_B, gpuCount > 1 ? streams[coli] : stream);
    	}
    	coliStart++;
	}
}


template<typename T> void CuMatrix<T>::transposeKernelPtr(DMatrix<T>& retD, void (*kernel)( T*, const T*,int,int)) {
	DMatrix<T>  d_A;
	tile0(d_A, lastMod == mod_host);
	transposeKernelPtrL(retD, kernel, d_A);
	invalidateHost();
}

template<typename T> void CuMatrix<T>::transposePitchKernelPtr(DMatrix<T>& retD, void (*kernel)( T*, const T*,int,int,int,int)) {
	DMatrix<T>  d_A;
	tile0(d_A, lastMod == mod_host);
	transposePitchKernelPtrL(retD, kernel, d_A);
	invalidateHost();
}

template<typename T> void CuMatrix<T>::transpose(DMatrix<T>& retD) {
	DMatrix<T>  d_A;
	tile0(d_A, lastMod == mod_host);
	transposeL(retD, d_A);
	invalidateHost();
}



#include "CuMatrixInster.cu"

