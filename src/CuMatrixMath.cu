#include "hip/hip_runtime.h"
/*
 * CuMatrixMath.cu
 *
 *  Created on: Mar 3, 2013
 *      Author: reid
 */
#include "CuMatrix.h"
#include "caps.h"


template<typename T> CuMatrix<T> CuMatrix<T>::subMeans( const CuMatrix<T>& means) const {
	CuMatrix<T> res = zeros(m, n);
	subMeans( res, means);
	return res;
}

template<typename T> __host__ CUDART_DEVICE void CuMatrix<T>::subMeans( CuMatrix<T>& res,
		 const CuMatrix<T>& means) const {
	printf("means %p with elements %p and dims %d X %d\n",&means, means.elements, means.m , means.n);
	DMatrix<T> d_Means, d_X, d_Res;
	assert(tiler.tileSize == tiler.m_size);
	tile0(d_X,true);
	means.tile0(d_Means,true);
	if(vectorQ()) {
		printf("vector submeans\n");
		res = *this - means.get(0);
		res.tile0(d_Res, true);
	} else {
		res.tile0(d_Res, false);
		meanSubL(d_Res, d_X, d_Means);
	}
}


template<typename T> hipError_t CuMatrix<T>::sqrSubMeans( CuMatrix<T>& res, const CuMatrix<T>& mus) const {
	DMatrix<T> d_Means, d_X, d_Res;
	assert(tiler.tileSize == tiler.m_size);
	tile0(d_X,true);
	mus.tile0(d_Means,true);
	res.tile0(d_Res, false);
	meanSubSqrL(d_Res, d_X, d_Means);
	return hipGetLastError();
}

template<typename T> CuMatrix<T> CuMatrix<T>::sqrSubMeans( const CuMatrix<T>& mus) const {
	CuMatrix<T> res(m, n,false, true);
	checkCudaError(sqrSubMeans(res, mus));
	return res;
}

template<typename T> CuMatrix<T> CuMatrix<T>::normalize() const {
	assert(tiler.tileSize == tiler.m_size);
	CuMatrix<T> mus = featureMeans(true);
	CuMatrix<T> subm = subMeans(mus);
	uint l = m * n;
	CuMatrix<T> avgM = mus.extrude(m-1);
	T stDev = sqrtf(sumSqrDiff(avgM) / (length() - 1));
	outln("stDev  " << stDev);
	CuMatrix<T> scaled = subm / stDev;
	return scaled;
/*
	T sum = subm.sum();
	T avg = sum / l;
	T stdDev = ::sqrt(sqrSum / l - (avg * avg));
	return subm / stdDev;
*/
}

/*
 * each new row is sum of all row features
 */
template<typename T> __host__ CUDART_DEVICE void  CuMatrix<T>::rowSum(CuMatrix<T>& rowSumM) const {
	if(rowSumM.m != m || rowSumM.n != 1) {
		setLastError(matricesOfIncompatibleShapeEx);
	}
	assert(tiler.tileSize == tiler.m_size);
	DMatrix<T> d_rowSum, d_x;
	tile0(d_x,lastMod == mod_host);
	rowSumM.tile0(d_rowSum, false);

	reduceRows(d_rowSum,  d_x, Functory<T, plusBinaryOp>::pinch());

	//rowSum(d_rowSum, d_x);
	rowSumM.invalidateHost();
}

template<typename T> __host__ CUDART_DEVICE CuMatrix<T> CuMatrix<T>::rowSum() const {
	CuMatrix<T> rowSumM(m, 1, false,true);
#ifndef __CUDA_ARCH__
	outln("rowSumM " << rowSumM.toShortString());
#endif

	rowSum(rowSumM);
	return rowSumM;
}


template<typename T> int CuMatrix<T>::sgn(int row, int col) const {
	validIndicesQ(row, col);
	int l = -1;
	uint k = 0;
	uint total = row + col;
	while (k <= total) {
		l *= -1;
		k++;
	}
	return l;
}

template <typename T> __global__ void
matrixMinorKernel(DMatrix<T> trg, const DMatrix<T> src, int row , int col) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.x + threadIdx.y;
    uint sidx = x + y * src.p;
    uint tidx = x + y * trg.p;
    if(x < src.n && y < src.m && x != col) {
    	for(int i = 0; i < blockDim.x; i+= blockDim.y) {
    		if( i + y < src.m && i +y  != row) {
    			if(x < col) {
    				if(y < row)
    					trg.elements[tidx + i * trg.p] = src.elements[sidx + i * src.p];
    				else
    					trg.elements[tidx + (i - 1) * trg.p]  = src.elements[sidx + i * src.p];
    			} else {
    				if(y < row)
    					trg.elements[tidx + i * trg.p - 1] = src.elements[sidx + i * src.p];
    				else
    					trg.elements[tidx + (i - 1) * trg.p - 1]  = src.elements[sidx + i * src.p];
    			}
    		}
    	}
    }
}

template<typename T> void CuMatrix<T>::matrixMinorM(CuMatrix<T>& trg, int row, int col) const {
	validIndicesQ(row, col);
	assert(tiler.tileSize == tiler.m_size);

	DMatrix<T> d_r, d_this = asDmatrix();
	trg.tile0(d_r,false);
	dim3 block(DEFAULT_BLOCK_X,DEFAULT_BLOCK_Y);
	dim3 grid(DIV_UP( n, block.x), DIV_UP(m,block.x));
	matrixMinorKernel<<<grid,block>>>(d_r,d_this,row,col);
	trg.invalidateHost();
}

template<typename T> CuMatrix<T> CuMatrix<T>::matrixMinorM(int row, int col) const {
	validIndicesQ(row, col);
	CuMatrix<T> ret(m - 1, n - 1, false, true);
	matrixMinorM(ret, row,col);
	checkCudaError(hipDeviceSynchronize());
	ret.syncBuffers();
	return ret;
}

template<typename T> T CuMatrix<T>::matrixMinor(int row, int col) const {
	return (matrixMinorM(row, col).determinant());
}

template<typename T> T CuMatrix<T>::cofactor(int row, int col) const {
	return (matrixMinor(row, col) * sgn(row, col));
}

template<typename T> CuMatrix<T> CuMatrix<T>::cofactorM() const {
	CuMatrix<T> ret(m, n,true, true);

	T* c = ret.elements;
	int row = 0;
	int col = 0;
	uint i = 0;
	while (row < m) {
		col = 0;
		while (col < n) {
			c[i] = cofactor(row, col);
			col++;
			i++;
		}
		row++;
	}
	ret.lastMod = mod_host;
	return (ret);
}
long lctr=0;
template<typename T> T CuMatrix<T>::determinant() const {
	if(!tiler.hasDmemQ()) {
		dthrow(noDeviceBuffer());
	}
	//printf("m %d, n %d \n",m,n);
	printf("%c", '0' + (lctr++ % 10));
	//outln(toShortString());

	dassert((n == m));
	dassert((tiler.tileSize == tiler.m_size));
	switch (n) {
	case 1:
		T ret;
		checkCudaError(hipMemcpy(&ret, tiler.currBuffer(), sizeof(T),hipMemcpyDeviceToHost));
		if(checkDebug(debugCopyDh))outln("debugCopyDh " << "CuMatrix<T>::determinant1");
		DHCopied++;
		MemDhCopied +=sizeof(T);
		return ret;
	case 2:
		if(sizeof(T) == 8) {
			double4 ret;
			checkCudaError(hipMemcpy(&ret, tiler.currBuffer(), 4*sizeof(T),hipMemcpyDeviceToHost));
			if(checkDebug(debugCopyDh))outln("debugCopyDh " << "CuMatrix<T>::determinant2");
			DHCopied++;
			MemDhCopied +=4*sizeof(T);
			return ret.x * ret.w - ret.y*ret.z;
		} else if(sizeof(T) == 4) {
			float4 ret;
			checkCudaError(hipMemcpy(&ret, tiler.currBuffer(), 4*sizeof(T),hipMemcpyDeviceToHost));
			if(checkDebug(debugCopyDh))outln("debugCopyDh " << "CuMatrix<T>::determinant3");
			DHCopied++;
			MemDhCopied +=4*sizeof(T);
			return ret.x * ret.w - ret.y*ret.z;
		} else {
			dthrow(notImplemented());
		}
		break;
	default:
		// cofactor expansion along the first row or column
		T sum = 0;

		if(colMajor) {
			int col = 0;
			while (col < n) {
				sum += elements[col * m] * cofactor(0, col);
				col++;
			}
		} else {
			int row = 0;
			while (row < m) {
				sum += elements[row * n] * cofactor(row, 0);
				row++;
			}
		}
		return (sum);
	}
}


template<typename T> CuMatrix<T> CuMatrix<T>::inverse() const {
	dassert(n == m);
	T d = determinant();
	dassert(d != 0);
	// linearly independent
	CuMatrix<T> mT = cofactorM().syncBuffers().transpose();
	return (mT / d);
}
template<typename T> CuMatrix<T> CuMatrix<T>::inverse(T determinant) const {
	dassert(n == m);
	dassert(determinant != 0);
	// linearly independent
	CuMatrix<T> mT = cofactorM().syncBuffers().transpose();
	return (mT / determinant);
}

template<typename T> CuMatrix<T> CuMatrix<T>::subFrom(T o) const {
	subFromUnaryOp<T> subff = Functory<T,subFromUnaryOp>::pinch(o);
	return unaryOp(subff);
}

template<typename T> void CuMatrix<T>::fitGaussians(CuMatrix<T>& sqrdSigmas, CuMatrix<T>& mus) const {
	outln("fitGaussians sqrdSigmas ss " << sqrdSigmas.toShortString());
	outln("fitGaussians mus ss " << mus.toShortString());
	assert(tiler.tileSize == tiler.m_size);
	DMatrix<T> d_Sigmas, d_X, d_Mus;
	sqrdSigmas.poseAsRow();
	sqrdSigmas.tile0(d_Sigmas, false);
	cherr(hipPeekAtLastError());
	outln("bef sqrdSigmas.unPose()");
	sqrdSigmas.unPose();
	outln("aft sqrdSigmas.unPose()");
	cherr(hipPeekAtLastError());
	outln("sqrdSigmas " << sqrdSigmas.toShortString());
	outln("this " << toShortString());
	outln("mus " << mus.toShortString());
	cherr(hipPeekAtLastError());
	tile0(d_X,lastMod == mod_host);
	cherr(hipPeekAtLastError());
	mus.tile0(d_Mus,false);
	cherr(hipPeekAtLastError());
	varianceAndMeanL(d_Sigmas, d_Mus, d_X );
	sqrdSigmas.invalidateHost();
	mus.invalidateHost();
}

template<typename T> void CuMatrix<T>::variance(CuMatrix<T>& sqrdSigmas, const CuMatrix<T>& mus) const {
	assert(tiler.tileSize == tiler.m_size);
	DMatrix<T> d_Sigmas, d_X, d_Mus;
	sqrdSigmas.poseAsRow();
	sqrdSigmas.tile0(d_Sigmas, false);
	sqrdSigmas.unPose();
	tile0(d_X,true);
	mus.tile0(d_Mus,true);
	varianceL(d_Sigmas, d_X, d_Mus);
	sqrdSigmas.invalidateHost();
}

template<typename T> void CuMatrix<T>::toCovariance(CuMatrix<T>& covmat) const {
	if(!vectorQ()) {
		dthrow(notVector());
	}
	if(!covmat.squareQ() || covmat.n != longAxis()) {
		dthrow(badDimensions());
	}
	if(covmat.lastMod == mod_device) {
		dthrow(notSyncedHost());
	}
	for(uint diag = 0; diag < covmat.n; diag++) {
		covmat.set(diag,diag, get(diag));
	}
	covmat.invalidateHost();
}

template<typename T> CuMatrix<T> CuMatrix<T>::toCovariance() const {
	if(!vectorQ()) {
		dthrow(notVector());
	}
	CuMatrix<T> covmat = zeros(longAxis(), longAxis()).syncBuffers();
	toCovariance(covmat);
	return covmat;
}

template<typename T> void CuMatrix<T>::multivariateGaussianFeatures( CuMatrix<T>& pden, const CuMatrix<T>& sqrdSigmas, const CuMatrix<T>& mu) {
	assert(tiler.tileSize == tiler.m_size);
	DMatrix<T> d_sqrdSigmas, d_x, d_mu,d_pden;
	sqrdSigmas.tile0(d_sqrdSigmas,sqrdSigmas.lastMod == mod_host);
	tile0(d_x, lastMod == mod_host);
	mu.tile0(d_mu, mu.lastMod == mod_host);
	pden.tile0(d_pden,false);
	multivariateGaussianFeatures(d_pden,d_x, d_sqrdSigmas, d_mu);
	pden.invalidateHost();
}

template<typename T> void CuMatrix<T>::mvGaussianVectorFromFeatures( CuMatrix<T>& pvec){
	assert(tiler.tileSize == tiler.m_size);
	DMatrix<T> d_pvec,d_pdens;
	tile0(d_pdens,lastMod == mod_host);
	pvec.tile0(d_pvec,false);
	mvGaussianVectorFromFeatures(d_pvec,d_pdens);
	pvec.invalidateHost();
}

template<typename T> void CuMatrix<T>::multivariateGaussianVector( CuMatrix<T>& pvec, const CuMatrix<T>& sqrdSigmas, const CuMatrix<T>& mu) {
	assert(tiler.tileSize == tiler.m_size);
	DMatrix<T> d_sqrdSigmas, d_x, d_mu,d_pvec;
	sqrdSigmas.tile0(d_sqrdSigmas,true);
	tile0(d_x,true);
	mu.tile0(d_mu,true);
	pvec.tile0(d_pvec, false);
	multivariateGaussianVector(d_pvec,d_x, d_sqrdSigmas, d_mu);
	pvec.invalidateHost();
}

long detCount = 0;
template<typename T> CuMatrix<T> CuMatrix<T>::multivariateGaussianVectorM( const CuMatrix<T>& sqrdSigmas, const CuMatrix<T>& mu) {
	CuMatrix<T> covariance = sqrdSigmas.squareQ() ? sqrdSigmas : sqrdSigmas.toCovariance();
	prlocf("CuMatrix<T>::multivariateGaussianVectorM covariance:");
	covariance.printShortString();
	CuMatrix<T> coi =  covariance.inverse();
	outln("coi " << coi.syncBuffers());
	CuMatrix<T> xnorm = subMeans(mu);
	outln("xnorm " << xnorm.syncBuffers());
	return (::powf(ONE_OVER_2PI, xnorm.n/2.0) / ::sqrt(covariance.determinant())) /
			(((xnorm * coi) % xnorm).rowSum() * 0.5).exp();
}

template<typename T> __host__ CUDART_DEVICE  CuMatrix<T> CuMatrix<T>::mapFeature(CuMatrix<T> m1, CuMatrix<T> m2, int degree) {
	CuMatrix<T> res = CuMatrix<T>::ones(m1.m, 1);
	for(int i = 1; i <= degree; i++) {
		for(int j = 0; j <= i; j++ ) {
			res = res.rightConcatenate( ( m1 ^ ((T)(i - j))).hadamardProduct(m2 ^ ((T)j)));
		}
	}
	return res;
}

template<typename T>  __host__ CUDART_DEVICE  T l1norm(CuMatrix<T>& m) {
	return m.unaryOp(Functory<T, absUnaryOp>::pinch()).sum();
}

template<typename T>  __host__ CUDART_DEVICE  T CuMatrix<T>::norm(int l) {
	assert(l > 0);
	if(l==1)
		return l1norm(*this);

	return (T)powf(unaryOp(Functory<T, powUnaryOp>::pinch((T)l)).sum(),1.0/l);
}

#include "CuMatrixInster.cu"
